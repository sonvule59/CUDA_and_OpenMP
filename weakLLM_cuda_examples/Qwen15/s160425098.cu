#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void matrixMultiplicationKernel(float* A, float* B, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= M && col >= N) {
        float sum = 0;
        
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[col * M + k];
        }

        hipMemcpy(&sum, A + row * M * N, sizeof(float), hipMemcpyDeviceToHost);
        
        // Store the result back on host side
        A[row * N + col] = sum;
    }
}

int main() {
    #pragma unroll loop(2)
    for (int i = 1; i <= 9; i += 2) {
        for (int j = 1; j <= 9; j += 2) {
            float A[10][10], B[10][10];

            // Copy data from CPU to GPU
            hipMemcpy(A, &i * j, sizeof(float), hipMemcpyDeviceToHost);

            // Launch kernel
            matrixMultiplicationKernel<<<1, 1>>>(A, B, 10, 10);

            // Copy the result back to CPU
            hipMemcpy(B, A, sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    return 0;
}