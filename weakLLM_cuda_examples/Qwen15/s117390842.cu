#include "hip/hip_runtime.h"
__global__ void add(int* a, int* b, int* c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= n) return;
    
    c[index] = a[index] + b[index];
    
    // Add padding to ensure correct size of output array
    c[index] += n - index;
    
    // Update global variables
    blockDim.x /= n;
    gridDim.x = n;
    
    // Compute sum of elements in each block
    int sum = 0;
    FOR(block, 0, blockDim.x, blockDim.y) {
        sum += c[block];
    }
    
    // Copy result to output array
    hipMemcpy(c, &sum, sizeof(int), hipMemcpyHostToDevice);
}

int main() {
    int n = 9;
    int a[n], b[n], c[n];
    
    // Allocate memory on device
    add<<<gridDim, blockDim>>>(a, b, c, n).transferToHost();
    
    // Print results
    FOR(i, 0, n, blockDim.x) {
        FOR(j, 0, n, blockDim.y) {
            cout << i + 1 << "x" << j + 1 << "=" << a[i] + b[j] << "\n";
        }
    }
    
    // Release memory on device
    hipDeviceSynchronize();
    
    return 0;
}