#include "hip/hip_runtime.h"
__global__ void multiply(int *A, int *B, int *C) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < A.size() && j < B.size()) {
        C[i] = A[i] * B[j];
    }
}