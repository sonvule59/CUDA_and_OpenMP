#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void kernel(int a, int b, int kake) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id < 1 || thread_id >= 10) return;

    int x = threadIdx.x;
    int y = threadIdx.y;
    int z = threadIdx.z;

    // Calculate the product of x and y for each element in the array
    int product_x = x * y;
    int product_y = x * z;
    int product_z = y * z;

    // Store the product in the kake variable
    kake += product_x * product_y * product_z;
}

int main() {
    int a = 0;
    int b = 0;
    int kake;

    // Allocate memory on the GPU
    hipMalloc(&a, sizeof(int));
    hipMalloc(&b, sizeof(int));
    hipMalloc(&kake, sizeof(int));

    // Set up the kernel function
    kernel<<<gridDim[0], blockDim[1]>>>(a, b, kake);

    // Launch the kernel on all threads
    hipMemcpy(kake, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kake, &b, sizeof(int), hipMemcpyHostToDevice);
    kernel<<<gridDim[0], blockDim[1]>>>(a, b, kake);

    // Read the result from the GPU
    int result = hipMemcpy(kake, &b, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    cout << "Result: " << result << endl;

    // Free the memory allocated on the GPU
    hipFree(a);
    hipFree(b);
    hipFree(kake);

    return 0;
}