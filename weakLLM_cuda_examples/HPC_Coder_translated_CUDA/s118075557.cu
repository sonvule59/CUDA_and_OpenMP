
#include <hip/hip_runtime.h>
#include <stdio.h>

#define rep(i,n) for(i=0;i<n;i++)

__global__ void kernel(int *a, int *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        printf("%dx%d=%d\n",i+1,i+1,(i+1)*(i+1));
    }
}

int main() {
    int i, k;
    int *a, *b;
    int n = 9;
    hipMalloc((void **)&a, n * sizeof(int));
    hipMalloc((void **)&b, n * sizeof(int));
    rep(i, n) {
        a[i] = i + 1;
        b[i] = i + 1;
    }
    kernel<<<1, n>>>(a, b, n);
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    return 0;
}