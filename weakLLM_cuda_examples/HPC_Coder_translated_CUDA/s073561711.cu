
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void multiply(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

int main() {
    int a[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int b[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int c[9];

    multiply<<<1, 9>>>(a, b, c);

    for (int i = 0; i < 9; i++) {
        printf("%d x %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}