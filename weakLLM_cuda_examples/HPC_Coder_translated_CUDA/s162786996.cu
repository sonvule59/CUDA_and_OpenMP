
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void print_table(int start, int end) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= start && idx <= end) {
		for (int i = 1; i <= 9; i++) {
			for (int j = 1; j <= 9; j++) {
				printf("%d x %d = %d\n", i, j, i*j);
			}
		}
	}
}

int main() {
	int start = 1;
	int end = 9;
	int blockSize = 25;
	int numBlocks = (end - start + 1 + blockSize - 1) / blockSize;
	print_table<<<numBlocks, blockSize>>>(start, end);
	hipDeviceSynchronize();
	return 0;
}