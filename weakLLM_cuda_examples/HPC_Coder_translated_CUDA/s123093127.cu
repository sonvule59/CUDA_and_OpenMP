
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <climits>
#include <cfloat>
#include <map>
#include <utility>
#include <set>
#include <iostream>
#include <memory>
#include <string>
#include <vector>
#include <algorithm>
#include <functional>
#include <sstream>
#include <complex>
#include <stack>
#include <queue>
using namespace std;
static const double EPS = 1e-5;
typedef long long ll;
#define FOR(i,n) for(i=0;i<n;i++)
#define rep(i,n) for(int i=0;i<n;i++)

__global__ void kernel(int *a, int *b, int *c){
	int i = threadIdx.x;
	a[i] = i;
	b[i] = i;
	c[i] = i;
}

int main(){
	int *a, *b, *c;
	hipMalloc((void **)&a, 9*sizeof(int));
	hipMalloc((void **)&b, 9*sizeof(int));
	hipMalloc((void **)&c, 9*sizeof(int));
	kernel<<<1,9>>>(a, b, c);
	hipDeviceSynchronize();
	rep(i,9){
		printf("%dx%d=%d\n",i+1,i+1,(i+1)*(i+1));
	}
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}