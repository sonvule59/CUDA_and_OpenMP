
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *m, int *n, int *q) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 100) {
		q[idx] = m[idx] * n[idx];
	}
}

int main() {
	int m[100], n[100], q[100];

	for(int i=0; i<100; i++) {
		m[i] = i+1;
		n[i] = i+1;
	}

	multiply<<<1,100>>>(m, n, q);

	for(int i=0; i<100; i++) {
		printf("%d x %d = %d\n", m[i], n[i], q[i]);
	}

	return 0;
}