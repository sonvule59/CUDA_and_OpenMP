#include "hip/hip_runtime.h"
c++
__global__ void add(int* a, int* b, int* c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < N) {
        c[index] += a[index] + b[index];
    }
}

int main() {

    int N = 9;
    int* a = new int[N];
    int* b = new int[N];
    int* c = new int[N];

    // Copy input values into arrays
    for (int i = 0; i < N; ++i) {
        a[i] = i + 1;
        b[i] = i + 2;
        c[i] = i + 3;
    }

    __shared__ int sharedSum = 0;

    add<<<16, 8>>>(a, b, c, N); // blockDim.x = 16, gridDim.x = 8

    // Access shared sum variable
    sharedSum = __syncthreads();

    for (int i = 0; i < N; ++i) {
        c[i] += sharedSum;
    }

    // Print final result
    std::cout << "Final Result: " << c[0] << " x " << c[1] << " x " << c[2] << "=" << sharedSum << std::endl;

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
 c++
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < N) {
        c[index] += a[index] + b[index];
    }
}

int main() {

    int N = 9;
    int* a = new int[N];
    int* b = new int[N];
    int* c = new int[N];

    // Copy input values into arrays
    for (int i = 0; i < N; ++i) {
        a[i] = i + 1;
        b[i] = i + 2;
        c[i] = i + 3;
    }

    __shared__ int sharedSum = 0;

    add<<<16, 8>>>(a, b, c, N); // blockDim.x =