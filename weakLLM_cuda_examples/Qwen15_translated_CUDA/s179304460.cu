#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel(int x, int y) {
    int sum = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < blockDim.x * gridDim.x; i += blockDim.x * blockDim.y) {
        sum += x * y;
    }
    printf("Sum of %dx%d is %d\n", x, y, sum);
}

int main() {
    int gridDimX = 9, blockDimY = 9;
    int numBlocksX = gridDimX / blockDimX, numBlocksY = gridDimY / blockDimY;
    
    // Allocate memory on GPU
    hipMalloc(&sumGridX, numBlocksX * blockDimX * sizeof(int));
    hipMalloc(&sumGridY, numBlocksY * blockDimY * sizeof(int));

    // Copy input data to GPU
    hipMemcpy(sumGridX, &gridData[0], numBlocksX * blockDimX * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(sumGridY, &gridData[1], numBlocksY * blockDimY * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on GPU
    kernel<<<gridDimX, blockDimY>>>(sumGridX, sumGridY);

    // Read result from GPU
    hipMemcpy(&result, sumGridX, numBlocksX * blockDimX * sizeof(int), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(sumGridX);
    hipFree(sumGridY);
    
    return 0;
}