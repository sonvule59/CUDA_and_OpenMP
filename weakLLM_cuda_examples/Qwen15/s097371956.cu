#include "hip/hip_runtime.h"
__global__ void printMatrix(int rows, int cols, int value) {
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadID < rows && threadID < cols) {
        // Print element at current grid location
        printf(" %dx%d=%d\n", threadID, row, col, value);
    }
}