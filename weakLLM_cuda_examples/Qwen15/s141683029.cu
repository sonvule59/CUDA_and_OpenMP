#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void my_kernel(int a, int b, int c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < 10) {
        // Access element of matrix A at row and column tid
        int x = a[tid];
        
        // Multiply matrix A by B
        int product = a[tid] * b;
        
        // Store product in matrix C
        c[tid] = product;
    }
}

int main() {
    int a = 1, b = 1, c = 0;
    
    // Allocate memory for matrix A on GPU
    hipMalloc((void**)&a, sizeof(int) * 10);
    
    // Allocate memory for matrix B on GPU
    hipMalloc((void**)&b, sizeof(int) * 10);
    
    // Allocate memory for matrix C on GPU
    hipMalloc((void**)&c, sizeof(int) * 10);
    
    // Copy data from CPU to GPU
    hipMemcpy(a, &a[0], sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b, &b[0], sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c, &c[0], sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on GPU
    my_kernel<<<10, 1>>>(a, b, c);

    // Copy result from GPU to CPU
    hipMemcpy(c, &c[0], sizeof(int), hipMemcpyDeviceToHost);

    // Free memory allocated on GPU
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}