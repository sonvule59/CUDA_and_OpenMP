#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void my_kernel(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < 9) {
        c[tid] = a[tid] * b[tid];
    }
}

int main() {
    int a[9], b[9], c[9];
    // Copy input values into GPU memory
    hipMemcpy(a, &a[0], sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b, &b[0], sizeof(int), hipMemcpyHostToDevice);

    // Set kernel block size and grid size
    dim3 block_size(16, 16, 16); // 16 threads per block, 16 blocks per grid
    dim3 grid_size(16, 16, 16); // 16 threads per grid

    // Create thread block for each grid element
    my_kernel<<<grid_size, block_size>>>(a, b, c);

    // Wait for all threads to finish
    cudaDeviceWaitAll();

    // Read output values from GPU memory
    int sum = 0;
    for (int i = 0; i < 9; i++) {
        sum += c[i];
    }

    printf("Sum of elements on the GPU: %d\n", sum);

    return 0;
}