
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int minNum, int maxNum) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= minNum && i <= maxNum && j >= minNum && j <= maxNum) {
    printf("%d x %d = %d\n", i, j, i * j);
  }
}

int main(void) {
  const int minNum = 1;
  const int maxNum = 9;

  int blockSize = 16;
  int numBlocks = (maxNum - minNum + 1) / blockSize;

  print_table<<<numBlocks, blockSize>>>(minNum, maxNum);

  hipDeviceSynchronize();

  return 0;
}