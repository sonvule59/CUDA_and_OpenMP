#include "hip/hip_runtime.h"
#include <stdio.h>

#define FOR(i,a,b) for(int i=(int)a;i<(int)b;++i)
#define REP(i,n) FOR(i,0,n)

__global__ void kernel(int *a, int *b)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < 10 && j < 10) {
    printf("%dx%d=%d\n",i,j,i*j);
  }
}

int main()
{
  int a[10][10];
  int b[10][10];
  REP(i,10) REP(j,10) {
    a[i][j] = i*j;
    b[i][j] = i*j;
  }
  int *d_a, *d_b;
  hipMalloc((void **)&d_a, 10*10*sizeof(int));
  hipMalloc((void **)&d_b, 10*10*sizeof(int));
  hipMemcpy(d_a, a, 10*10*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, 10*10*sizeof(int), hipMemcpyHostToDevice);
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((10+threadsPerBlock.x-1)/threadsPerBlock.x, (10+threadsPerBlock.y-1)/threadsPerBlock.y);
  kernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b);
  hipMemcpy(a, d_a, 10*10*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, d_b, 10*