
#include <hip/hip_runtime.h>
__global__ void print_table(int *table) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < 10 && j < 10) {
		printf("%d x %d = %d\n", i, j, i*j);
	}
}

int main() {
	int *table;
	hipMalloc((void **)&table, 100 * sizeof(int));
	print_table<<<dim3(10, 10), dim3(16, 16)>>>(table);
	hipDeviceSynchronize();
	hipFree(table);
	return 0;
}