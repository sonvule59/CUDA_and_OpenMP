#include "hip/hip_runtime.h"
__global__ void printGridKernel(float* grid, int rows, int cols) {
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadID < rows && threadID < cols) {
        float x = grid[threadID];
        float y = grid[threadID+1];
        float z = grid[threadID+2];

        // Print the grid at this point
        printf("x = %f, y = %f, z = %f\n", x, y, z);
    }
}

int main() {
    float grid[9][9] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}, {0, 1, 2}, {3, 4, 5}, {6, 7, 8}, {9, 0, 1}, {2, 3, 4}, {5, 6, 7}};
    int rows = sizeof(grid) / sizeof(grid[0]);
    int cols = sizeof(grid[0]) / sizeof(grid[0][0]);

    __shared__ float sharedGrid[row * cols];

    printGridKernel<<<grid_size, block_size>>>(sharedGrid, rows, cols);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", sharedGrid[i * cols + j]);
        }
        printf("\n");
    }

    return 0;
}