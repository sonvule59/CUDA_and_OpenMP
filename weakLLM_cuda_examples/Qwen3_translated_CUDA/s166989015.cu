#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <cstring>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <vector>
#include <stack>
#include <queue>
#include <deque>
#include <map>
#include <list>
#include <set>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <bitset>
#include <functional>

__global__ void kernel(int *a, int *b, int *c) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;
    int x = a[i * 3 + j * 3 + k * 3];
    int y = b[i * 3 + j * 3 + k * 3];
    int z = c[i * 3 + j * 3 + k * 3];
    int x1 = a[i * 3 + j * 3 + k * 3];
    int y1 = b[i * 3 + j * 3 + k * 3];
    int z1 = c[i * 3 + j * 3 + k * 3];
    int x2 = a[i * 3 + j * 3 + k * 3];
    int y2 = b[i * 3 + j * 3 + k * 3];
    int z2 = c[i * 3 + j * 3 + k * 3];
    int x3 = a[i * 3 + j * 3 + k * 3];
    int y3 = b[i * 3 + j * 3 + k * 3];
    int z3 = c[i * 3 + j * 3 + k * 3];
    int x4 = a[i * 3 + j * 3 + k * 3];
    int y4 = b[i * 3 + j * 3 + k * 3];
    int z4 = c[i * 3 + j * 3 + k * 3];
    int x5 = a[i * 3 + j * 3 + k * 3];
    int y5 = b[i * 3 + j * 3 + k * 3];
    int z5 = c[i * 3 + j * 3 + k * 3];
    int x6 = a[i * 3 + j * 3 + k * 3];
    int y6