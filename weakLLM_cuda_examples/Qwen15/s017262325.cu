#include "hip/hip_runtime.h"
__global__ void printMatrix(int i, int j, int value) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id < 1 || thread_id > 9)
        return;

    // Check if the row and column indices are within the range of the matrix
    if(thread_id >= i && thread_id <= i*9 && thread_id < j && thread_id <= j*9)
        value[thread_id] = value[thread_id] * value[thread_id];
}