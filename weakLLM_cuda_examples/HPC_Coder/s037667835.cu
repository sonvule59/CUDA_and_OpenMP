
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void print_table(int *table) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 100) {
        int i = idx / 10;
        int j = idx % 10;
        printf("%d x %d = %d\n", i, j, i*j);
    }
}

int main() {
    int *table;
    hipMalloc((void **)&table, 100 * sizeof(int));
    print_table<<<1, 100>>>(table);
    hipDeviceSynchronize();
    hipFree(table);
    return 0;
}