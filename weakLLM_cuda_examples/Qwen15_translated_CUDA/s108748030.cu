#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void kernel(int x, int y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate product of x and y on GPU
    float product = x * y;

    // Store product in local memory
    hipMemsetAsync(&product, 0, sizeof(float), hipMemcpyHostToDevice);

    // Update global variables
    x = x + 1;
    y = y + 1;

    // Execute kernel on CPU
    kernel<<<gridDim.x, blockDim.y>>>(i, j);
    
    // Update global variables again
    x = x - 1;
    y = y - 1;

    // Copy local memory back to host
    hipMemcpy(host_ptr, &product, sizeof(float), hipMemcpyDeviceToHost);

    // Release device memory
    hipMemsetAsync(&product, 0, sizeof(float), hipMemcpyDeviceToHost);
}

int main(int argc, const char * argv[]) {
    // Initialize CUDA runtime
    cudaInit();

    // Create grid with dimensions (9x9)
    dim3 gridDim(9, 9);
    dim3 blockDim(gridDim.x, blockDim.y);

    // Allocate shared memory for product variable
    cudaMallocSync(&shared_ptr, sizeof(float) * 9 * 9);

    // Set block size to (1x1)
    blockDim.set(1, 1);

    // Create kernel function
    kernel<<<gridDim.x, blockDim.y>>>(0, 0);

    // Launch kernel on GPU
    hipLaunchKernel(reinterpret_cast<const void*>(kernel), gridDim.x, gridDim.y, 0, shared_ptr);

    // Wait for kernel to finish executing
    cudaWaitAll();

    // Clean up CUDA resources
    cudaFreeSync(shared_ptr);
    cudaFreeMemSync(shared_ptr);

    // Print result
    cout << "Product of x and y on GPU: " << static_cast<float>(shared_ptr[0]) << endl;

    return 0;
}