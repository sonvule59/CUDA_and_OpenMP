#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel(int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < 9 && j < 9) {
		int a = i + 1;
		int b = j + 1;
		c[i * 9 + j] = a * b;
	}
}

int main() {
	int a[9][9];
	int b[9][9];
	int c[9][9];

	for (int i = 0; i < 9; ++i) {
		for (int j = 0; j < 9; ++j) {
			a[i][j] = i + 1;
			b[i][j] = j + 1;
		}
	}

	kernel<<<dim3(3, 3), dim3(3, 3)>>>(a, b, c);

	for (int i = 0; i < 9; ++i) {
		for (int j = 0; j < 9; ++j) {
			printf("%dx%d=%d\n", a[i][j], b[i][j], c[i][j]);
		}
	}

	return 0;
}