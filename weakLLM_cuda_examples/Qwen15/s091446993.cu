#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void my_kernel(float* A, float* B, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < N) {
        float sum = 0.0f;
        for(int i = tid; i < N; i += blockDim.x * gridDim.x) {
            sum += A[i] * B[tid];
        }
        printf("Sum at thread %d: %f\n", tid, sum);
    }
}

int main(int argc, char const *argv[]) {
    // Load the input data into GPU memory
    float* A = (float*)malloc(N * sizeof(float));
    VINT(A,N);
    
    // Create a kernel function that performs the multiplication operation on A and B
    float* B = (float*)malloc(N * sizeof(float));
    VINT(B,N);
    my_kernel<<<1,1>>>(A,B,N);

    // Copy the output data back to host memory
    hipMemcpy(B,A,N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU
    int num_blocks = 5;
    int blocks_per_grid = 2;
    int grid_size = N / num_blocks;
    int threads_per_block = blockDim.x * grid_size;
    int device_size = grid_size * blockDim.x * blockDim.x;
    int num_threads = num_blocks * threads_per_block;
    int block_start = blockDim.x * blockIdx.x + threadIdx.x;
    int block_end = blockDim.x * blockIdx.x + blockDim.x * blockDim.x * num_threads - threads_per_block;
    my_kernel<<<block_start,block_end,num_threads>>>(B,A,N);

    // Read the result from host memory
    float* result = (float*)malloc(N * sizeof(float));
    VINT(result,N);
    hipMemcpy(result,B,N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for(int i = 0; i < N; i++) {
        printf("Result at thread %d: %f\n", i, result[i]);
    }

    free(A);
    free(B);
    free(result);

    return 0;
}