#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void kernel(int x, int y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= 8 && j <= 8) {
        // Calculate product of two numbers
        int product = x * y;

        // Store result in GPU memory
        hipMemcpy(&product, &product, sizeof(int), hipMemcpyHostToDevice);

        // Update global variables
        i += blockDim.x * gridDim.x;
        j += blockDim.y * gridDim.y;
    }
}

int main() {
    int i = 1, ii = 1;
    
    dim3 threads(2, 2, 2); // 2 threads per block, 2 blocks per grid
    dim3 grid(5, 5, 5); // 5x5 grid
    
    kernel<<<grid, threads>>>(i, ii);
    
    // Read results from GPU memory
    int product[10];
    hipMemcpy(product, kernel.getOutput(), sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    for (int j = 0; j < 10; j++) {
        printf("%d ", product[j]);
    }

    return 0;
}