
#include <hip/hip_runtime.h>
__global__ void kernel(float *A, float *B, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid<N) {
        float sum = 0;
        for(int i=0; i<N; i+= blockDim.x) {
            sum += A[i] * B[tid];
        }
        printf("Sum at thread %d: %f\n", tid, sum);
    }
}