
#include <hip/hip_runtime.h>
__global__ void matrix_multiplication(int m, int n, int *A, int *B, int *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m && i < n) {
        int sum = 0;
        
        for (int j = 0; j < n; j++) {
            sum += A[i] * B[j];
        }
        
        C[i] = sum;
    }
}