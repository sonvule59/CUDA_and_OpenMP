
#include <hip/hip_runtime.h>
__global__ void matrixMultiplicationKernel(float* A, float* B, int N, float* C) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= N && col >= N) {
        float sum = 0.0f;
        
        // Loop over each element of A and B
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                sum += A[i] * B[j];
            }
        }

        // Store the result in C
        C[row * N + col] = sum;
    }
}