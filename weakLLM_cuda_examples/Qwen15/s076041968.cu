#include "hip/hip_runtime.h"
__global__ void multiplyKernel(int* a, int* b, int c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < 1 || tid >= 10) {
        return;
    }

    int rowA = a[tid];
    int colB = b[tid];

    // Perform multiplication on rowA and columnB
    int product = rowA * colB;

    // Store the result in the local memory
    c[tid] = product;
}