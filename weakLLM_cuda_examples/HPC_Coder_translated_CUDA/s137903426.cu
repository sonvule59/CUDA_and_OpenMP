
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *x, int *y, int *z) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 10) {
        for (int i = 0; i < 10; i++) {
            *x = *x + 1;
            *y = *x * *z;
            printf("%d x %d = %d\n", *x, *z, *y);
            *z = *z + 1;
            if (*z >= 9) {
                break;
            }
        }
        *z = 0;
    }
}

int main() {
    int x, y, z;
    x = 0;
    y = 0;
    z = 0;
    int *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, sizeof(int));
    hipMalloc((void **)&d_y, sizeof(int));
    hipMalloc((void **)&d_z, sizeof(int));
    hipMemcpy(d_x, &x, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, &y, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_z, &z, sizeof(int), hipMemcpyHostToDevice);
    kernel<<<1, 10>>>(d_x, d_y, d_z);
    hipMemcpy(&x, d_x, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&y, d_y, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&z, d_z, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}