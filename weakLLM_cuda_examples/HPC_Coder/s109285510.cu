
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *m, int *n, int *result) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 100) {
		result[idx] = m[idx] * n[idx];
	}
}

int main(int argc, char *argv[]) {
	int m[100], n[100], result[100];
	for (int i = 0; i < 100; i++) {
		m[i] = i + 1;
		n[i] = i + 1;
	}

	int *d_m, *d_n, *d_result;
	hipMalloc((void **)&d_m, 100 * sizeof(int));
	hipMalloc((void **)&d_n, 100 * sizeof(int));
	hipMalloc((void **)&d_result, 100 * sizeof(int));

	hipMemcpy(d_m, m, 100 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, n, 100 * sizeof(int), hipMemcpyHostToDevice);

	multiply<<<1, 100>>>(d_m, d_n, d_result);

	hipMemcpy(result, d_result, 100 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 100; i++) {
		printf("%d x %d = %d\n", m[i], n[i], result[i]);
	}

	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_result);

	return 0;
}