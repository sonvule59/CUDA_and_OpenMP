
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void multiply(int *a, int *b, int *c, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n) {
		c[i] = a[i] * b[i];
	}
}

int main() {
	int n = 10;
	int *a, *b, *c;
	a = (int *)malloc(n * sizeof(int));
	b = (int *)malloc(n * sizeof(int));
	c = (int *)malloc(n * sizeof(int));

	for (int i = 0; i < n; i++) {
		a[i] = i;
		b[i] = i;
	}

	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, n * sizeof(int));
	hipMalloc((void **)&d_b, n * sizeof(int));
	hipMalloc((void **)&d_c, n * sizeof(int));

	hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

	multiply<<<1, n>>>(d_a, d_b, d_c, n);

	hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++) {
		printf("%d * %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(a);
	free(b);
	free(c);

	return 0;
}