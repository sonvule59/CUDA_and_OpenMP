
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int start, int end) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= start && i <= end) {
        for (int n = 1; n <= 9; n++) {
            printf("%d x %d = %d\n", i, n, i * n);
        }
    }
}

int main() {
    int start = 1;
    int end = 9;
    int blockSize = 256;
    int numBlocks = (end - start + 1) / blockSize;
    print_table<<<numBlocks, blockSize>>>(start, end);
    hipDeviceSynchronize();
    return 0;
}