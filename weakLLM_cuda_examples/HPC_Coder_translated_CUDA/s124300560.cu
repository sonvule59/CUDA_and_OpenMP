
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_multiplication_table(int start, int end){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= start && i <= end){
        for(int j=1;j<10;j++){
            printf("%dx%d=%d\n",i,j,i*j);
        }
    }
}

int main(void){
    int start = 1;
    int end = 10;
    int threads_per_block = 256;
    int blocks_per_grid = (end - start + 1 + threads_per_block - 1) / threads_per_block;
    print_multiplication_table<<<blocks_per_grid, threads_per_block>>>(start, end);
    hipDeviceSynchronize();
}