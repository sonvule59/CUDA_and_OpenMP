
#include <hip/hip_runtime.h>
#include <stdio.h>

#define REP(i, n) for(int i=0;i<(int)(n);i++)
#define ALL(x) (x).begin(),(x).end()
const int INF = 1e9;

__global__ void kernel(int *a, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        printf("%d x %d = %d\n", idx+1, idx+1, (idx+1) * (idx+1));
    }
}

int main() {
    int n = 9;
    int *a;
    hipMalloc((void **)&a, n * sizeof(int));
    kernel<<<1, n>>>(a, n);
    hipDeviceSynchronize();
    hipFree(a);
    return 0;
}