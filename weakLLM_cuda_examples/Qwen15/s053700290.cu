
#include <hip/hip_runtime.h>
__global__ void matrixMultiplicationKernel(int n, int m, float* A, float* B) {
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadID < n && threadID < m){
        float sum = 0.0f;
        
        // loop through each element of A and B
        for(int i = threadIdx.x; i < n; i += blockDim.x)
            for(int j = threadIdx.x; j < m; j += blockDim.x){
                sum += A[i] * B[j];
            }

        // store the result in a local variable
        float c = sum;
        
        // update the global variable with the result
        A[threadID] = c;
        B[threadID] = c;
    }
}