#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;

#define for_(i,a,b) for(int i=(a);i<(b);++i)

__global__ void print_table(int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		for_(j,1,10) {
			cout << i << "x" << j << "=" << i * j << endl;
		}
	}
}

int main() {
	int n = 10;
	print_table<<<1,n>>>(n);
	hipDeviceSynchronize();
}