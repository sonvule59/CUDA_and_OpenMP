#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void multiply(int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < 10 && j < 10) {
		c[i * 10 + j] = a[i] * b[j];
	}
}

int main(int argc, char const *argv[])
{
	int a[10][10], b[10][10], c[10][10];
	int i, j;
	for(i=0;i<10;i++) {
		for(j=0;j<10;j++) {
			a[i][j] = i+1;
			b[i][j] = j+1;
		}
	}

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((10+threadsPerBlock.x-1)/threadsPerBlock.x, (10+threadsPerBlock.y-1)/threadsPerBlock.y);
	multiply<<<numBlocks, threadsPerBlock>>>(a, b, c);

	for(i=0;i<10;i++) {
		for(j=0;j<10;j++) {
			printf("%dx%d=%d\n", i+1, j+1, c[i][j]);
		}
	}
	return 0;
}