
#include <hip/hip_runtime.h>
__global__ void kernel(int i, int r, int num) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id < r) {
        int temp = num * thread_id;
        printf("thread %d x %d = %d\n", thread_id, r, temp);
    }
}