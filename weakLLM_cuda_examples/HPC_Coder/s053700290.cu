
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

__global__ void print_multiplication_table() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i <= 9 && j <= 9) {
        printf("%d x %d = %d\n", i, j, i * j);
    }
}

int main() {
    int blockSize = 16;
    int numBlocks = 9;
    print_multiplication_table<<<numBlocks, blockSize>>>();
    hipDeviceSynchronize();
    return EXIT_SUCCESS;
}