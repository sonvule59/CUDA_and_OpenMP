
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void print_table(int *table){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < 9 && j < 9){
		printf("%d x %d = %d\n", i + 1, j + 1, (i+1)*(j+1));
	}
}

int main(){
	int table[9][9];
	for (int i = 0; i < 9; i++){
		for (int j = 0; j < 9; j++){
			table[i][j] = (i+1)*(j+1);
		}
	}

	int *d_table;
	hipMalloc((void **)&d_table, 9*9*sizeof(int));
	hipMemcpy(d_table, table, 9*9*sizeof(int), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(9, 9);
	dim3 numBlocks((9+threadsPerBlock.x-1)/threadsPerBlock.x, (9+threadsPerBlock.y-1)/threadsPerBlock.y);
	print_table<<<numBlocks, threadsPerBlock>>>(d_table);

	hipDeviceSynchronize();
	hipFree(d_table);

	return 0;
}