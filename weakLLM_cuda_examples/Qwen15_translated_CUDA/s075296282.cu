#include "hip/hip_runtime.h"
__global__ void multiplyArrays(int* arrA, int* arrB, int size) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id < size){
        int product = arrA[thread_id] * arrB[thread_id];
        __syncthreads();
        if(thread_id % 2 == 0){
            // If both threads have even number of elements, add them together
            int sum = product + arrA[thread_id+1] + arrB[thread_id+2];
            __syncthreads();
            // Update the product with the sum
            arrA[thread_id] = product;
            arrB[thread_id] = sum;
        } else {
            // If one thread has odd number of elements, subtract it from the other
            int difference = product - arrA[thread_id-1] - arrB[thread_id-2];
            __syncthreads();
            // Update the product with the difference
            arrA[thread_id] = product;
            arrB[thread_id] = difference;
        }
    }
}

int main() {
    int size = 9;
    int* arrA = new int[size];
    int* arrB = new int[size];

    // Initialize arrays with random values
    for (int i = 0; i < size; i++) {
        arrA[i] = rand() % 100;
        arrB[i] = rand() % 100;
    }

    // Multiply arrays on GPU
    multiplyArrays(arrA, arrB, size);

    // Print the result
    for (int i = 0; i < size; i++) {
        std::cout << arrA[i] << " x " << arrB[i] << "=" << arrA[i] * arrB[i] << std::endl;
    }

    delete[] arrA;
    delete[] arrB;

    return 0;
}