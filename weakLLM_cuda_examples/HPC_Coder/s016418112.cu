
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void print_table(int p, int n)
{
    printf("%dx%d=%d\n",p,n,p*n);
}
int main()
{
    int p,n;
    for(p=1;p<=9;p++)
    {
    for(n=1;n<10;n++)
    {
        print_table<<<1,1>>>(p,n);
    }
    }
    return 0;
}