
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void multiply(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

int main() {
    int *a, *b, *c;
    int size = 9;

    // Allocate memory for arrays on the GPU
    hipMalloc((void **)&a, size * sizeof(int));
    hipMalloc((void **)&b, size * sizeof(int));
    hipMalloc((void **)&c, size * sizeof(int));

    // Copy data from host to device
    for (int i = 0; i < size; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    // Launch the CUDA kernel
    multiply<<<1, size>>>(a, b, c);

    // Copy data from device to host
    for (int i = 0; i < size; i++) {
        printf("%d x %d = %d\n", a[i], b[i], c[i]);
    }

    // Free memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}