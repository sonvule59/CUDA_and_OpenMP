#include "hip/hip_runtime.h"
__global__ void kernel(int x, int y) {
    int sum = 0;
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < blockDim.x * gridDim.x; i += blockDim.x * blockDim.y) {
        sum += x * y;
    }
    
    // Store result in device memory
    int z = sum;
    
    // Update global variable
    global_x += blockDim.x * gridDim.x;
    global_y += blockDim.y * gridDim.x;
    
    // Write result back to host memory
    hipMemcpy(host_x, &z, sizeof(z), hipMemcpyDeviceToHost);
}

int main() {
    dim3 threads_per_block(3, 3, 3); // 3 threads per block, 3 blocks in grid
    dim3 grid_size(9, 9, 9); // 9x9x9 grid
    
    // Allocate device memory
    int* host_x = new int[grid_size.x * grid_size.y * blockDim.x * blockDim.y];
    
    // Launch kernel on GPU
    kernel<<<threads_per_block, grid_size>>>(host_x, host_x + grid_size.x * grid_size.y * blockDim.x * blockDim.y);
    
    // Copy result back to host memory
    hipMemcpy(host_x, host_x + grid_size.x * grid_size.y * blockDim.x * blockDim.y, sizeof(int), hipMemcpyDeviceToHost);
    
    // Print results
    for (int i = 0; i < grid_size.x; i++) {
        for (int j = 0; j < grid_size.y; j++) {
            cout << host_x[i * grid_size.y + j] << " ";
        }
        cout << endl;
    }

    delete[] host_x;
    
    return 0;
}