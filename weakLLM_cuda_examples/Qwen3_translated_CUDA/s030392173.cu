#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void __kernel__(int *a, int *b, int *c)
{
    int idx = threadIdx.x;
    int i = idx;
    int j = idx;
    int k = i * j;
    a[idx] = k;
    b[idx] = k;
    c[idx] = k;
}

int main()
{
    int *a = new int[10];
    int *b = new int[10];
    int *c = new int[10];
    for(int i=0; i<10; i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = i;
    }
    __kernel<<<0, 1>>>(a, b, c);
    printf("Result:\n");
    for(int i=0; i<10; i++)
    {
        printf("%d %d %d\n", a[i], b[i], c[i]);
    }
    return 0;
}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}

}