#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void matrix multiplication_kernel(float* A, float* B, float* C, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row >= m && col >= n) {
        float sum = 0.0f;
        for(int k = 0; k < n; k++) {
            sum += A[row*m+k] * B[col*n+k];
        }

        C[row*m+col] = sum;
    }
}

int main() {
    // Initialize CUDA runtime
    cudaRuntimeInit();

    // Allocate memory for input matrices A and B
    float* A = (float*)malloc(sizeof(float)*m);
    float* B = (float*)malloc(sizeof(float)*n);

    // Copy input matrices to CUDA memory
    hipMemcpy(A, &A[m], sizeof(float)*m, hipMemcpyHostToDevice);
    hipMemcpy(B, &B[n], sizeof(float)*n, hipMemcpyHostToDevice);

    // Set block size and grid size
    int blockDimX = 32;
    int blockDimY = 32;
    int numBlocks = ceil(m / blockDimX) + ceil(n / blockDimY);

    // Create a kernel function for matrix multiplication
    matrix multiplication_kernel<<<numBlocks, blockDimX, blockDimY>>>(A, B, C, m, n);

    // Launch kernel on GPU
    int threads_per_block = blockDim.x * blockDim.y;
    int grid_size = blockDimX * blockDimY;
    int num_threads = threads_per_block * numBlocks;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cudaGetDeviceID(prop.major, prop.minor, 0);

    // Create a CUDA context
    cudaContextCreate(&context);

    // Launch kernel on GPU with device context
    hipLaunchKernel(reinterpret_cast<const void*>(multiplication_kernel), num_threads, grid_size, threads_per_block, A, B, C, &context);

    // Free memory allocated for input matrices
    free(A);
    free(B);

    // Clean up CUDA resources
    hipDeviceSynchronize();
    hipFree(context);

    return 0;
}