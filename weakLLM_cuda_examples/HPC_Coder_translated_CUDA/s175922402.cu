
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void multiply(int *a, int *b, int *c){
	int i = threadIdx.x;
	c[i] = a[i] * b[i];
}

int main(){
	int a[10], b[10], c[10];
	for(int i=0;i<10;i++){
		a[i] = i+1;
		b[i] = i+1;
	}
	
	multiply<<<1,10>>>(a,b,c);
	
	for(int i=0;i<10;i++){
		printf("%d x %d = %d\n",a[i],b[i],c[i]);
	}
	
	return 0;
}