#include "hip/hip_runtime.h"
__global__ void __global_func ( int *a, int *b, int *c )
{
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;

  if ( i == 0 && j == 0 && k == 0 )
  {
    c[0] = a[0] * b[0];
  }
  else if ( i == 0 && j == 0 && k != 0 )
  {
    c[k] = a[0] * b[k];
  }
  else if ( i == 0 && j != 0 && k == 0 )
  {
    c[j] = a[j] * b[0];
  }
  else if ( i == 0 && j != 0 && k != 0 )
  {
    c[k] = a[j] * b[k];
  }
  else if ( i == 1 && j == 0 && k == 0 )
  {
    c[0] = a[1] * b[0];
  }
  else if ( i == 1 && j == 0 && k != 0 )
  {
    c[k] = a[1] * b[k];
  }
  else if ( i == 1 && j != 0 && k == 0 )
  {
    c[j] = a[1] * b[0];
  }
  else if ( i == 1 && j != 0 && k != 0 )
  {
    c[k] = a[j] * b[k];
  }
  else if ( i == 2 && j == 0 && k == 0 )
  {
    c[0] = a[2] * b[0];
  }
  else if ( i == 2 && j == 0 && k != 0 )
  {
    c[k] = a[2] * b[k];
  }
  else if ( i == 2 && j != 0 && k == 0 )
  {
    c[j] = a[2] * b[0];
  }
  else if ( i == 2 && j != 0 && k != 0 )
  {
    c[k] = a[j] * b[k];
  }
  else if ( i == 3 && j == 0 && k == 0 )
  {
    c[0] = a[3] * b