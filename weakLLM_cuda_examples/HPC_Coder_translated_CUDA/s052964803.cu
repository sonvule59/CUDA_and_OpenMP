
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void print_table(int *table) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < 81) {
		printf("%d x %d = %d\n", i / 9 + 1, i % 9 + 1, (i/9+1)*(i%9+1));
	}
}

int main() {
	int table[81];
	for (int i = 0; i < 81; i++) {
		table[i] = i;
	}

	print_table<<<1, 81>>>(table);

	return 0;
}