
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void print_table(int *table){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < 100)
        printf("%d x %d = %d\n", idx/10, idx%10, table[idx]);
}
int main(){
    int table[100];
    for(int i=0;i<100;i++)
        table[i] = i+1;
    int *d_table;
    hipMalloc((void **)&d_table, 100*sizeof(int));
    hipMemcpy(d_table, table, 100*sizeof(int), hipMemcpyHostToDevice);
    print_table<<<1,100>>>(d_table);
    hipDeviceSynchronize();
    hipFree(d_table);
    return 0;
}