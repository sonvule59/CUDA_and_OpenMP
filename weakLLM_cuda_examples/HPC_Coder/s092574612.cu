
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

#define FOR(i,l,r) for(int i = (l);i < (r);i++)
#define ALL(x) (x).begin(),(x).end()
template<typename T> bool chmax(T& a,const T& b){return a < b ? (a = b,true) : false;}
template<typename T> bool chmin(T& a,const T& b){return b < a ? (a = b,true) : false;}
typedef long long ll;

__global__ void kernel(int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 100)
	{
		a[idx] = idx * idx;
		b[idx] = idx * idx * idx;
	}
}

int main()
{
	int *a, *b;
	hipMalloc((void **)&a, 100 * sizeof(int));
	hipMalloc((void **)&b, 100 * sizeof(int));

	kernel<<<1,100>>>(a, b);

	hipDeviceSynchronize();

	for(int i = 0;i < 100;i++)
	{
		printf("%d^2=%d\n",i,a[i]);
		printf("%d^3=%d\n",i,b[i]);
	}

	hipFree(a);
	hipFree(b);

	return 0;
}