
#include <hip/hip_runtime.h>
__global__ void matrix_multiplication_kernel(int rows, int cols, int* A, int* B, int* C) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row >= rows && col >= cols) {
    int sum = 0;
    
    // Add elements of A and B column-wise
    for (int i = 0; i < cols; ++i) {
      sum += A[row * cols + i] * B[col * cols + i];
    }

    // Store result in C
    C[row * cols + col] = sum;
  }
}