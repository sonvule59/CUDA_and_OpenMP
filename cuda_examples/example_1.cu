#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

// Device function that does the padding
__device__ void paddingChar(unsigned char *P, uint64_t dataBlockSize, uint64_t paddingSize) {
    P[dataBlockSize] = 0x80;
    for (int i = 1; i <= paddingSize - 9; i++) {
        P[dataBlockSize + i] = 0x00;
    }
    for (int i = 1; i <= 8; i++) {
        P[dataBlockSize + paddingSize - i] = (unsigned char)((8 * dataBlockSize) >> ((i - 1) * 8));
    }
}

// CUDA kernel: each thread applies padding to one data block
__global__ void paddingKernel(unsigned char *P, uint64_t *dataBlockSizes, uint64_t *paddingSizes, int numBlocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numBlocks) {
        unsigned char *blockPtr = P + idx * (dataBlockSizes[idx] + paddingSizes[idx]);
        paddingChar(blockPtr, dataBlockSizes[idx], paddingSizes[idx]);
    }
}

int main() {
    const int numBlocks = 1;
    uint64_t dataBlockSize = 56;
    uint64_t paddingSize = 64 - dataBlockSize;
    size_t totalSize = dataBlockSize + paddingSize;

    // Host memory
    unsigned char *h_P = new unsigned char[totalSize];
    uint64_t *h_dataBlockSizes = new uint64_t[numBlocks];
    uint64_t *h_paddingSizes = new uint64_t[numBlocks];

    h_dataBlockSizes[0] = dataBlockSize;
    h_paddingSizes[0] = paddingSize;

    // Fill initial data
    for (int i = 0; i < dataBlockSize; i++) {
        h_P[i] = (unsigned char)(i + 1);
    }

    // Device memory
    unsigned char *d_P;
    uint64_t *d_dataBlockSizes, *d_paddingSizes;

    hipMalloc(&d_P, totalSize);
    hipMalloc(&d_dataBlockSizes, sizeof(uint64_t) * numBlocks);
    hipMalloc(&d_paddingSizes, sizeof(uint64_t) * numBlocks);

    // Copy host to device
    hipMemcpy(d_P, h_P, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_dataBlockSizes, h_dataBlockSizes, sizeof(uint64_t) * numBlocks, hipMemcpyHostToDevice);
    hipMemcpy(d_paddingSizes, h_paddingSizes, sizeof(uint64_t) * numBlocks, hipMemcpyHostToDevice);

    // Launch kernel
    paddingKernel<<<1, 1>>>(d_P, d_dataBlockSizes, d_paddingSizes, numBlocks);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_P, d_P, totalSize, hipMemcpyDeviceToHost);

    // Print result
    printf("Padded Block:\n");
    for (int i = 0; i < totalSize; i++) {
        printf("%02x ", h_P[i]);
    }
    printf("\n");

    // Cleanup
    delete[] h_P;
    delete[] h_dataBlockSizes;
    delete[] h_paddingSizes;
    hipFree(d_P);
    hipFree(d_dataBlockSizes);
    hipFree(d_paddingSizes);

    return 0;
}


// #include <cuda_runtime.h>
// #include <stdint.h>

// __device__ void paddingChar(unsigned char *P, uint64_t dataBlockSize, uint64_t paddingSize) {
//     // Step 1: Set the first byte after the data block to 0x80
//     P[dataBlockSize] = 0x80;

//     // Step 2: Fill with 0x00 up to paddingSize - 9
//     for (int i = 1; i <= paddingSize - 9; i++) {
//         P[dataBlockSize + i] = 0x00;
//     }

//     // Step 3: Append the 64-bit length of the input in bits at the end
//     for (int i = 1; i <= 8; i++) {
//         P[dataBlockSize + paddingSize - i] =
//             (unsigned char)((8 * dataBlockSize) >> (8 * (i - 1)));
//     }
// }

// __global__ void paddingCharKernel(unsigned char *P, uint64_t dataBlockSize, uint64_t paddingSize) {
//     // Only use one thread here since padding is done sequentially
//     if (threadIdx.x == 0 && blockIdx.x == 0) {
//         paddingChar(P, dataBlockSize, paddingSize);
//     }
// }

// int main() {
//     uint64_t dataBlockSize = 64;     // example size
//     uint64_t paddingSize = 64;       // example padding

//     // Allocate host memory
//     unsigned char *h_P = new unsigned char[dataBlockSize + paddingSize]{};

//     // Allocate device memory
//     unsigned char *d_P;
//     cudaMalloc(&d_P, dataBlockSize + paddingSize);
//     cudaMemcpy(d_P, h_P, dataBlockSize + paddingSize, cudaMemcpyHostToDevice);

//     // Launch kernel
//     paddingCharKernel<<<1, 1>>>(d_P, dataBlockSize, paddingSize);
//     cudaDeviceSynchronize();

//     // Copy result back to host
//     cudaMemcpy(h_P, d_P, dataBlockSize + paddingSize, cudaMemcpyDeviceToHost);

//     // Free memory
//     cudaFree(d_P);
//     delete[] h_P;

//     return 0;
// }
