#include "hip/hip_runtime.h"
__global__ void kernel(int a, int b) {
    int n = blockDim.x * gridDim.x;
    int i = threadIdx.x + blockIdx.x * n;
    int j = threadIdx.y + blockIdx.y * n;

    if (i < 1 || i > 9 || j < 1 || j > 9)
        return;

    int temp = a * b;
    a = temp / b;
    b = temp % b;

    __syncthreads();

    if (j < 1 || j > 9)
        return;

    a += a * j;
    b += b * j;

    __syncthreads();

    if (i < 2 || i > 8)
        return;

    a /= 2;
    b /= 2;

    __syncthreads();

    if (j < 2 || j > 7)
        return;

    a += a * j;
    b += b * j;

    __syncthreads();
}

int main() {
    int a = 1, b = 1, i, n;

    for (i = 1; i <= 9; i++) {
        for (n = 1; n <= 9; n++) {
            kernel<<<grid_size, block_size>>>(a, b);
            // Print the result on the GPU
            std::cout << "a = " << a << ", b = " << b << std::endl;
            b = n + 1;
        }
        a = i + 1;
        b = 1;
    }

    return 0;
}