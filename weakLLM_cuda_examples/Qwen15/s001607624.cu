#include <hip/hip_runtime.h>

__global__ void matrixMultiplicationKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < N) {
        float sum = 0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 1024;  // Matrix size (N x N)
    size_t size = N * N * sizeof(float);

    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}