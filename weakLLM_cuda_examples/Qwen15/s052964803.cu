#include "hip/hip_runtime.h"
__global__ void print_i32(int* arr, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        int index = tid * blockDim.x + threadIdx.y;
        int value = arr[index];
        
        // Add some padding for alignment
        value += blockDim.x * gridDim.x * 4;
        
        // Write the result to a file
        FILE* file = fopen("output.txt", "w");
        fprintf(file, "%d ", value);
        fclose(file);
    }
}