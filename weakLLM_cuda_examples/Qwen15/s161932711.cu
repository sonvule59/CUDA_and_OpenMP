#include "hip/hip_runtime.h"
c
#include <hip/hip_runtime.h>

__global__ void matrixMultiplicationKernel(int x, int y, int z, int w) {
    int threadIdxX = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIdxY = threadIdx.y + blockIdx.y * blockDim.y;
    int threadIdxZ = threadIdx.z + blockIdx.z * blockDim.z;

    // Check if the threads belong to the same block
    if (threadIdxX >= 0 && threadIdxX < blockDim.x &&
        threadIdxY >= 0 && threadIdxY < blockDim.y &&
        threadIdxZ >= 0 && threadIdxZ < blockDim.z) {

        // Calculate the product of x and y for each thread
        int productX = x * y;
        
        // Store the product in a local variable
        int product = productX;

        // Update the product on the grid
        blockDim.x = blockDim.x * 2;
        blockDim.y = blockDim.y * 2;
        blockDim.z = blockDim.z * 2;
        gridDim.x = blockDim.x / 2;
        gridDim.y = blockDim.y / 2;
        gridDim.z = blockDim.z / 2;
        threadIdx.x = threadIdx.x - blockDim.x * 2;
        threadIdx.y = threadIdx.y - blockDim.y * 2;
        threadIdx.z = threadIdx.z - blockDim.z * 2;

        // Update the product on the grid
        blockDim.x = blockDim.x * 2;
        blockDim.y = blockDim.y * 2;
        blockDim.z = blockDim.z * 2;
        gridDim.x = blockDim.x / 2;
        gridDim.y = blockDim.y / 2;
        gridDim.z = blockDim.z / 2;
        threadIdx.x = threadIdx.x - blockDim.x * 2;
        threadIdx.y = threadIdx.y - blockDim.y * 2;
        threadIdx.z = threadIdx.z - blockDim.z * 2;

        // Update the product on the grid
        blockDim.x = blockDim.x * 2;
        blockDim.y = blockDim.y * 2;
        blockDim.z = blockDim.z * 2;
        gridDim.x = blockDim.x / 2;
        gridDim.y = blockDim.y / 2;
        gridDim.z = blockDim.z / 2;
        threadIdx.x = threadIdx.x - blockDim.x * 2;
        threadIdx.y = threadIdx.y - blockDim.y * 2;
        threadIdx.z = threadIdx.z - blockDim.z * 2;

        // Update the product on the grid