
#include <hip/hip_runtime.h>
__global__ void maxAreaKernel(float* A, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row >= rows && col >= cols)
    {
        float area = A[row * cols];
        
        // Check if the maximum value is within the grid boundaries
        if(area > 0)
        {
            // Update the maximum value on the GPU
            A[row * cols] = max(A[row * cols], area);

            // Print the updated value
            printf("Max Area at (%dx%d): %f\n", row, col, area);
        }
    }
}