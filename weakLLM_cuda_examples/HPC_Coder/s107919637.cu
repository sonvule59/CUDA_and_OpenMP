#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void multiply(int *a, int *b, int *c) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    c[i*blockDim.y + j] = a[i] * b[j];
}

int main() {
    int a[10][10];
    int b[10][10];
    int c[10][10];

    for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            a[i][j] = i+1;
            b[i][j] = j+1;
        }
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, 10*10*sizeof(int));
    hipMalloc((void **)&d_b, 10*10*sizeof(int));
    hipMalloc((void **)&d_c, 10*10*sizeof(int));

    hipMemcpy(d_a, a, 10*10*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 10*10*sizeof(int), hipMemcpyHostToDevice);

    multiply<<<10,10>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, 10*10*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            printf("%d x %d = %d\n", i+1, j+1, c[i][j]);
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d