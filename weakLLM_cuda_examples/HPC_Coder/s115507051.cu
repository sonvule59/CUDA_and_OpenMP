
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int *table) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < 100) {
    printf("%dx%d=%d\n", idx / 10, idx % 10, table[idx]);
  }
}

int main(void) {
  int table[100];
  for (int i = 0; i < 100; i++) {
    table[i] = (i / 10) * (i % 10) * (i / 10) * (i % 10);
  }

  int *d_table;
  hipMalloc((void **)&d_table, 100 * sizeof(int));
  hipMemcpy(d_table, table, 100 * sizeof(int), hipMemcpyHostToDevice);

  print_table<<<1, 100>>>(d_table);

  hipDeviceSynchronize();
  hipFree(d_table);

  return 0;
}