#include "hip/hip_runtime.h"
__global__ void add(x_t *x, y_t *y) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < 9) {
        x[tid] += y[tid];
    }
}