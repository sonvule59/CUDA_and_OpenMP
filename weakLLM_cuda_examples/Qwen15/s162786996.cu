#include "hip/hip_runtime.h"
__global__ void matrix multiplicationKernel(int n, int m, int k) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= n && col >= m && row < n && col < m) {
        int product = 0;
        
        for (int i = 0; i < m; i++) {
            product += threadIdx.x * threadIdx.y * blockDim.x * blockDim.y * blockDim.y * blockDim.y;
        }

        // Store the result in a device variable
        int* result = (int*)deviceMalloc(n * m * sizeof(int));
        hipMemcpy(result, &product, n * m * sizeof(int), hipMemcpyDeviceToHost);

        // Update the global variables
        globalMatrix[row][col] = result[k];
    }
}

// Global variables
vll globalMatrix[10][10];

int main() {
    // Launch the kernel on multiple threads
    dim3 gridDim(10, 10);
    dim3 threadGridDim(16, 16);
    
    __shared__ int sharedResult[10 * 10 * 10];
    
    // Initialize the shared memory with zeros
    hipMemcpy(sharedResult, globalMatrix.data(), globalMatrix.size() * sizeof(int), hipMemcpyDeviceToDevice);

    // Launch the kernel on multiple threads
    matrix multiplicationKernel<<<gridDim, threadGridDim>>>(10, 10, 1);
    
    // Wait for all threads to finish
    blockID = threadIdx.x + blockDim.x * blockIdx.x;
    while (blockID < 10 * 10 * 10) {
        blockID += blockDim.x * blockDim.y * blockDim.z;
    }
    
    // Extract the results from the shared memory
    int* result = sharedResult[threadID * blockDim.x + threadIdx.y * blockDim.z];
    
    // Print the results
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            cout << i << 'x' << j << '=' << result[i * 10 + j] << endl;
        }
    }
    
    // Free the shared memory
    deviceFree(sharedResult);
    
    return 0;
}