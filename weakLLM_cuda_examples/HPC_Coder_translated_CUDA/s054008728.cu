
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *x, int *y) {
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < 100) {
x[idx] = idx;
y[idx] = idx;
}
}

int main() {
int *x, *y;
int size = 100;
x = (int *)malloc(size * sizeof(int));
y = (int *)malloc(size * sizeof(int));

kernel<<<1, size>>>(x, y);

for (int i = 0; i < size; i++) {
printf("%d x %d = %d\n", x[i], y[i], x[i] * y[i]);
}

free(x);
free(y);
return 0;
}