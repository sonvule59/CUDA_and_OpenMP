#include "hip/hip_runtime.h"
__global__ void kernel(int i, int j) {
  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int local_i = thread_id * blockDim.x + threadIdx.y;
  int local_j = thread_id * blockDim.x + threadIdx.z;

  // Accessing elements of matrix A
  int a[3][3] = {{i, j, 0}, {i+1, j, 0}, {i, j+1, 0}};
  
  // Transposing matrix A
  int temp[3][3] = a;
  for (int k = 0; k < 3; k++) {
    for (int l = 0; l < 3; l++) {
      temp[k][l] = temp[l][k];
    }
  }

  // Adding element at local_i and j
  int sum = temp[thread_id][thread_id] + temp[thread_id+1][thread_id+1] + temp[thread_id][thread_id+2];

  // Store result in global variable
  int result = sum;
}

int main() {
  int gridDimX = 512;
  int blockDimY = 512;
  int gridSizeZ = 64;

  // Creating grid of threads
  dim3 threadsGrid(gridDimX, blockDimY, gridSizeZ);
  dim3 blockThreads(threadsGrid.x, threadsGrid.y, threadsGrid.z);

  // Launch kernel on each thread
  for (int i = 0; i < blockDimY; i++) {
    for (int j = 0; j < blockDimX; j++) {
      kernel<<<blockThreads>>>(i, j);
    }
  }

  // Collect results from all threads
  int globalSum = 0;
  for (int i = 0; i < blockDimY; i++) {
    for (int j = 0; j < blockDimX; j++) {
      globalSum += kernel[i][j];
    }
  }

  // Print final result
  cout << "Global Sum: " << globalSum << endl;

  return 0;
}