
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 10) {
        for (int i = 1; i <= 9; i++) {
            c[idx] = a[idx] * i;
            printf("%d x %d = %d\n", a[idx], i, c[idx]);
        }
    }
}

int main() {
    int a[10], b[10], c[10];
    for (int i = 0; i < 10; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
    }
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, 10 * sizeof(int));
    hipMalloc((void **)&d_b, 10 * sizeof(int));
    hipMalloc((void **)&d_c, 10 * sizeof(int));
    hipMemcpy(d_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 10 * sizeof(int), hipMemcpyHostToDevice);
    multiply<<<1, 10>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, 10 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++) {
        printf("%d x %d = %d\n", a[i], i + 1, c[i]);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}