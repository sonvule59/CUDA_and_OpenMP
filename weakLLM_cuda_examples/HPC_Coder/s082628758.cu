
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void multiply(int *a, int *b, int *c, int n, int m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<n && j<m)
	{
		c[i*m+j] = a[i]*b[j];
	}
}

int main()
{
	int *a, *b, *c;
	int n, m;
	scanf("%d %d", &n, &m);
	a = (int *)malloc(n*sizeof(int));
	b = (int *)malloc(m*sizeof(int));
	c = (int *)malloc((n*m)*sizeof(int));

	// Initialize a and b

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((n+threadsPerBlock.x-1)/threadsPerBlock.x, (m+threadsPerBlock.y-1)/threadsPerBlock.y);
	multiply<<<numBlocks, threadsPerBlock>>>(a, b, c, n, m);

	// Print the result

	free(a);
	free(b);
	free(c);
	return 0;
}