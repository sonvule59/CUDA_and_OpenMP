
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void print_table(int start, int end){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= start && i <= end){
		for(int j = 1 ; j <= 9 ; ++j)
		printf("%dx%d=%d\n",i,j,i*j);
	}
}

int main(){
	int start = 1, end = 9;
	print_table<<<1,10>>>(start, end);
	hipDeviceSynchronize();
	return 0;
}