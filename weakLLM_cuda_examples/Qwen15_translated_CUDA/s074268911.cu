#include "hip/hip_runtime.h"
__global__ void my_kernel(int n, int *s, int *g) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        s[i] = g * i;
    }
}