
#include <hip/hip_runtime.h>
#include <stdio.h>
int s,e;
__global__ void kernel(int *s, int *e) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 10) {
        s[idx] = (e[idx] = e[idx] % 9 + 1) < 2 ? s[idx] : 0;
        printf("%d x %d = %d\n", s[idx], e[idx], s[idx] * e[idx]);
    }
}
int main() {
    int s[10], e[10];
    for (int i = 0; i < 10; i++) {
        s[i] = 0;
        e[i] = i;
    }
    int *d_s, *d_e;
    hipMalloc((void **)&d_s, 10 * sizeof(int));
    hipMalloc((void **)&d_e, 10 * sizeof(int));
    hipMemcpy(d_s, s, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_e, e, 10 * sizeof(int), hipMemcpyHostToDevice);
    kernel<<<1, 10>>>(d_s, d_e);
    hipMemcpy(s, d_s, 10 * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_s);
    hipFree(d_e);
    return 0;
}