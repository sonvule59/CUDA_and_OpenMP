
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void printTable(int *table){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < 10){
		for(int jndex = 1 ; jndex <= 9 ; jndex++){
			printf("%d x %d = %d\n", index, jndex, index*jndex);
		}
	}
}

int main(){
	int *table;
	hipMalloc((void **)&table, sizeof(int)*10*10);
	printTable<<<1,10>>>(table);
	hipDeviceSynchronize();
	hipFree(table);
}