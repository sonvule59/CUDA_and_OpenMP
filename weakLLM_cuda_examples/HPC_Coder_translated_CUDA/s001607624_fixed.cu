#include <iostream>
#include <hip/hip_runtime.h>

__global__ void print_table() {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < 10 && col < 10) {
        int val = (row + 1) * (col + 1);
        printf("%d x %d = %d\n", row + 1, col + 1, val);
    }
}
int main() {
    dim3 threadsPerBlock(10, 10);
    dim3 numBlocks(1, 1);

    print_table<<<numBlocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    return 0;
}
