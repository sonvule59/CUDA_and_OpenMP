
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cstdlib>
#include <stack>
#include <queue>
#include <list>
#include <vector>
#include <sstream>

using namespace std;

__global__ void f_input(int *i, int *j)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 10)
    {
        for (int j = 1; j <= 9; ++j)
        {
            printf("%d x %d = %d\n", idx, j, idx * j);
        }
    }
}

int main()
{
    int *i, *j;
    hipMalloc((void **)&i, sizeof(int));
    hipMalloc((void **)&j, sizeof(int));

    f_input<<<1, 10>>>(i, j);

    hipDeviceSynchronize();

    hipFree(i);
    hipFree(j);

    return 0;
}