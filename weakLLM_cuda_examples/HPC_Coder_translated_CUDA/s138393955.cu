
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        int n1 = i + 1;
        int n2 = j + 1;
        printf("%d x %d = %d\n", n1, n2, n1 * n2);
    }
}

int main() {
    int n = 9;
    int threadsPerBlock = 16;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    print_table<<<blocksPerGrid, threadsPerBlock>>>(n);

    hipDeviceSynchronize();

    return 0;
}