
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__ void print_table(int y, int i){
    printf("%d x %d = %d\n", y, i, y*i);
}

int main(){
    int y,i;
    for(y = 1;y < 10;y++){
          for(i = 1;i < 10;i++){
          print_table<<<1,1>>>(y,i);
          }
    }
}