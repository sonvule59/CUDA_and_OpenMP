
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void print_table(int *table) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 100) {
        printf("%d x %d = %d\n", idx / 10, idx % 10, table[idx]);
    }
}

int main() {
    int *table;
    hipMallocManaged(&table, 100 * sizeof(int));
    for (int i = 1; i <= 9; i++) {
        for (int j = 1; j <= 9; j++) {
            table[i * 10 + j] = i * j;
        }
    }
    int blockSize = 10;
    int numBlocks = (100 + blockSize - 1) / blockSize;
    print_table<<<numBlocks, blockSize>>>(table);
    hipDeviceSynchronize();
    hipFree(table);
    return 0;
}