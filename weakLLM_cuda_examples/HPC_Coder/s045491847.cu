
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void multiply(int *a, int *b, int *c, int n){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		c[i] = a[i] * b[i];
	}
}

int main(){
	int a[9] = {1,2,3,4,5,6,7,8,9};
	int b[9] = {1,2,3,4,5,6,7,8,9};
	int c[9];
	int n = 9;

	multiply<<<1,n>>>(a,b,c,n);

	for(int i=0;i<n;i++){
		printf("%d x %d = %d\n",a[i],b[i],c[i]);
	}

	return 0;
}