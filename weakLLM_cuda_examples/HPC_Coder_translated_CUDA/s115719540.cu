#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *a, int *b) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	a[idx] = b[idx];
}

int main() {
	int *a, *b;
	int n = 10;
	a = (int *)malloc(n * sizeof(int));
	b = (int *)malloc(n * sizeof(int));

	REP(i, n) {
		a[i] = i;
		b[i] = i * i;
	}

	kernel<<<1, n>>>(a, b);

	REP(i, n) {
		printf("%d x %d = %d\n", a[i], a[i], b[i]);
	}

	free(a);
	free(b);

	return 0;
}