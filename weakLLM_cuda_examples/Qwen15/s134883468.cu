#include "hip/hip_runtime.h"
__global__ void print_matrix(int matrix[10][10], int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        for (int jndex = 1; jndex < size; jndex++) {
            cout << matrix[index] << " ";
        }
        cout << endl;
    }
}

int main() {
    int matrix[10][10];
    // Initialize matrix with some values
    matrix[0][0] = 1;
    matrix[0][1] = 2;
    matrix[0][2] = 3;
    matrix[0][3] = 4;
    matrix[0][4] = 5;
    matrix[0][5] = 6;
    matrix[0][6] = 7;
    matrix[0][7] = 8;
    matrix[0][8] = 9;
    matrix[0][9] = 10;

    int size = 10;
    // Print matrix on GPU using CUDA kernel
    print_matrix<<<gridDim=1, blockDim*x>>>(matrix, size);

    return 0;
}