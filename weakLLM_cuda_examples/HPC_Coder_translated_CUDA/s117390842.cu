
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define FOR(i,a,b) for(int i=(a);i<(b);i++)
#define REP(i,n) for(int i=0;i<(n);i++)

#define PI 3.14159265358479

__global__ void kernel(int *a, int *b){
	REP(i, 9){
		REP(j, 9){
			printf("%d x %d = %d\n", i + 1, j + 1, (i + 1)*(j + 1));
		}
	}
}

int main(){
	int *a, *b;
	int size = sizeof(int) * 9 * 9;
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	REP(i, 9){
		REP(j, 9){
			a[i * 9 + j] = i + 1;
			b[i * 9 + j] = j + 1;
		}
	}
	kernel<<<1, 1>>>(a, b);
	hipDeviceSynchronize();
	free(a);
	free(b);
	return 0;
}