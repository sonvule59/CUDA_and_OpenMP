
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int start, int end) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= start && i <= end) {
        for (int k = 1; k <= 9; k++) {
            printf("%dx%d=%d\n", i, k, i * k);
        }
    }
}

int main() {
    int start = 1;
    int end = 9;
    int blockSize = 256;
    int numBlocks = (end - start + 1) / blockSize;

    print_table<<<numBlocks, blockSize>>>(start, end);

    hipDeviceSynchronize();

    return 0;
}