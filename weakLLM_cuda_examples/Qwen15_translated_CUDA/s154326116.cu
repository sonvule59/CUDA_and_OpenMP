#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void QQ(int a, int b) {
    // Check if a is greater than or equal to 10
    if (a >= 10) {
        return;
    }

    // Check if b is less than or equal to 10
    if (b < 10) {
        printf("x%d x%d = %d\n", a, b, a * b);
        
        // Call QQ function on a larger block with b + 1
        QQ(a, b + 1);

        // Call QQ function again on a smaller block with a + 1
        QQ(a + 1, 1);
    } else {
        // If b is greater than or equal to 10, call QQ function on a block with a + 1
        QQ(a + 1, 1);
    }
}

int main() {
    // Create a CUDA device context
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // Get the number of threads per block
    int numBlocks = prop.multiProcessorCount / prop.deviceSize;

    // Allocate memory for the block sizes
    int blockSize[2] = {16, 32};

    // Initialize the block sizes
    hipMemcpy(blockSize, blockSize, sizeof(blockSize), hipMemcpyHostToDevice);

    // Create a kernel object
    kernel<<<numBlocks, 1>>>(blockSize, &QQ);

    // Launch the kernel
    kernelLaunch<<<numBlocks, 1>>>(blockSize, &QQ);

    // Read the results from the host
    int result[2];
    hipMemcpy(result, QQ.get(), sizeof(result), hipMemcpyDeviceToHost);

    // Print the results
    printf("QQ result:\n");
    for (int i = 0; i < 2; i++) {
        printf("%d ", result[i]);
    }

    return 0;
}