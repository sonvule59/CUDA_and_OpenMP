#include "hip/hip_runtime.h"
__global__ void kernel(int x, int y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < 10) {
        // Calculate product of y and i
        int product = y * i;
        
        // Store product in global memory
        globalStorage[i] = product;
    }
}