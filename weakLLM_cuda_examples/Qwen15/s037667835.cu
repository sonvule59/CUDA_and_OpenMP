#include "hip/hip_runtime.h"
__global__ void kernel(int x, int y) {
    int sum = 0;
    
    // Accessing elements of matrix A on GPU
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < blockDim.x * gridDim.x; i += blockDim.x * blockDim.y) {
        sum += x * y;
    }

    // Updating global variable 'sum'
    gl_FragColor[0] = sum / blockDim.x * 255.0;
}