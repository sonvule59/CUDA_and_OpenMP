
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void multiply(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}
int main() {
    int a[10], b[10], c[10];
    int n = 10;
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i;
    }
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);
    multiply<<<1, n>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        cout << a[i] << "x" << b[i] << "=" << c[i] << endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}