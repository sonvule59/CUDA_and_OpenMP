
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 10)
    {
        for (int j = 1; j <= 9; ++j)
        {
            c[i * 9 + j] = a[i] * b[j];
        }
    }
}

int main()
{
    int a[10], b[9], c[10];
    for (int i = 0; i < 10; ++i)
    {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, 10 * sizeof(int));
    hipMalloc((void **)&d_b, 9 * sizeof(int));
    hipMalloc((void **)&d_c, 10 * sizeof(int));

    hipMemcpy(d_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 9 * sizeof(int), hipMemcpyHostToDevice);

    multiply<<<1, 10>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, 10 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i)
    {
        for (int j = 1; j <= 9; ++j)
        {
            printf("%dx%d=%d\n", i + 1, j, c[i * 9 + j]);
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}