
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 10) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    int a[10], b[10], c[10];
    int *d_a, *d_b, *d_c;

    // Initialize arrays a and b
    for (int i = 0; i < 10; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    // Allocate memory on the GPU
    hipMalloc((void **)&d_a, 10 * sizeof(int));
    hipMalloc((void **)&d_b, 10 * sizeof(int));
    hipMalloc((void **)&d_c, 10 * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 10 * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    multiply<<<1, 10>>>(d_a, d_b, d_c);

    // Copy data from device to host
    hipMemcpy(c, d_c, 10 * sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print the result
    for (int i = 0; i < 10; i++) {
        printf("%d x %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}