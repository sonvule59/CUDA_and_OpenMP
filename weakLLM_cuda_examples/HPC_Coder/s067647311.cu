#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <set>
#include <sstream>

using namespace std;

__global__ void expr(int x, int y, string* c) {
    stringstream ss;
    ss << x << "x" << y << "=" << x*y;
    c[threadIdx.x] = ss.str();
}

int main() {
    set<string> c;
    for (int i=1e3; i; --i) {
        c.insert(expr(rand() % 9 + 1, rand() % 9 + 1));
    }
    for (auto it=c.begin(); it<c.end(); ++it) {
        cout << *it << endl;
    }
    return 0;
}