#include "hip/hip_runtime.h"
__global__ void main() {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Global block size and thread index
    int k = blockIdx.y * blockDim.y + threadIdx.y; // Global block size and thread index

    if (i <= 8 && k <= 8) { // Check if block is within range of threads
        for (int j = 1; j <= 9; j++) {
            float x = i * j; // Calculate element on local grid
            float y = k * j; // Calculate element on local grid
            printf("x%d y%d = %f\n", j, j, x * y); // Print element on local grid
        }
    }
}