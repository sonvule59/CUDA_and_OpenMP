
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

__global__ void kernel(int *a, int *b){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx<9){
		for(int c=0;c<9;c++)printf("%d*%d=%d\n",idx+1,c+1,(idx+1)*(c+1));
	}
}

int main(){
	int *a, *b;
	hipMalloc((void **)&a, sizeof(int));
	hipMalloc((void **)&b, sizeof(int));

	kernel<<<1,9>>>(a, b);

	hipFree(a);
	hipFree(b);

	return 0;
}