#include "hip/hip_runtime.h"
c
#include <hip/hip_runtime.h>

__global__ void kernel(int m, int n)
{
  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.z + blockIdx.z * blockDim.z;

  // Accessing elements of matrix using grid indices
  int element = (row * m * n) + (col * m);

  // Adding element to local matrix
  int local_matrix[row * m * n] = element + 1;

  // Updating global matrix
  for(int i = 0; i < m; ++i)
  {
    for(int j = 0; j < n; ++j)
    {
      global_matrix[i * n + j] = local_matrix[i * n + j];
    }
  }

  // Updating global matrix by multiplying it with a constant value
  global_matrix[0] *= 2;
}

int main()
{
  int m = 1;
  int n = 1;

  while (true)
  {
    if (n == 10)
    {
      m = m + 1;
      n = 1;
    }
    if (m == 10) break;

    // Launch kernel on GPU
    hipMalloc(&local_matrix, sizeof(int) * (m * n));
    hipMemcpy(local_matrix, &global_matrix[0], sizeof(int) * (m * n), hipMemcpyHostToDevice);
    kernel<<<gridDim, blockDim>>>(m, n);

    // Wait for all threads to finish execution
    cudaDeviceWaitIdle();

    // Accessing elements of matrix using grid indices
    int element = (threadIdx.x + blockIdx.x * blockDim.x) * blockDim.x + threadIdx.y + blockIdx.y * blockDim.y;
    int local_matrix_row = threadIdx.y + blockIdx.y * blockDim.y;
    int local_matrix_col = threadIdx.z + blockIdx.z * blockDim.z;

    // Adding element to local matrix
    int local_matrix[row * m * n] = element + 1;

    // Updating global matrix
    for(int i = 0; i < m; ++i)
    {
      for(int j = 0; j < n; ++j)
      {
        global_matrix[i * n + j] = local_matrix[i * n + j];
      }
    }

    // Updating global matrix by multiplying it with a constant value
    global_matrix[0] *=