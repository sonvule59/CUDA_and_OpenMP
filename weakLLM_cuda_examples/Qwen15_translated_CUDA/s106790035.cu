
#include <hip/hip_runtime.h>
__global__ void multKernel(int *A, int *B, int N, int M) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M) {
        A[i] *= B[j];
    }
}