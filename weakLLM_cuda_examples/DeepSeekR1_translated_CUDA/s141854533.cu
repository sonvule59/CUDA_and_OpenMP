#include "hip/hip_runtime.h"
__global__ sum = i * j;
cout << i << "x" << j << "=" << sum << endl;
__global__ sum = i * j;
cout << i << "x" << j << "=" << sum << endl;
</think>

#include<hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void my_function() {
    int i = __local__(int) i;
    int j = __local__(int) j;
    int sum = i * j;
    cout << i << "x" << j << "=" << sum << endl;
}

int main() {
    int sum;
    
    for(int i=1; i<10; i++){
        for(int j=1; j<10; j++){
            sum = i * j;
            cout << i << "x" << j << "=" << sum << endl;
        }
    }

    return 0;
}