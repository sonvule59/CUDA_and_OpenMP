#include "hip/hip_runtime.h"
c++
__global__ void addXZ(int *x, int *z) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;
    for (int i = 0; i < blockDim.x * gridDim.x; i += blockDim.x) {
        sum += x[idx] * z[idx];
    }
    // Store result in local memory
    *z = sum;
}

int main() {
    int x = 0, y = 0, z = 0;
    dim3 block_size(16, 16, 1); // 16x16x1 thread block size
    dim3 grid_size(16, 16, 1); // 16x16x1 grid size

    addXZ<<<grid_size, block_size>>>(x, &z);

    while (true) {
        x += 1;
        while (true) {
            z += 1;
            y = x * z;
            printf("%d %d=%d\n", x, z, y);
            if (z >= 9) {
                break;
            }
        }
        z = 0;
        if (x >= 9) {
            break;
        }
    }

    return 0;
}
 c++
__global__ void addXZ(int *x, int *z) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;
    for (int i = 0; i < blockDim.x * gridDim.x; i += blockDim.x) {
        sum += x[idx] * z[idx];
    }
    // Store result in global memory
    *z = sum;
}

int main() {
    int x = 0, y = 0, z = 0;
    dim3 block_size(16, 16, 1); // 16x16x1 thread block size
    dim3 grid_size(16, 16, 1); // 16x16x1 grid size

    addXZ<<<grid_size, block_size>>>(x, &z);

    while (true) {
        x += 1;
        while (true) {
            z += 1;
            y = x * z;
            printf("%d %d=%d\n", x, z, y);
            if (z >=