#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

__global__ void multiply(int *a, int *b, int *c, int n) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		c[idx] = a[idx] * b[idx];
	}
}

int main() {
	int n1, n2;
	int *a, *b, *c;

	// Allocate memory on the GPU
	hipMalloc((void **)&a, n1 * sizeof(int));
	hipMalloc((void **)&b, n2 * sizeof(int));
	hipMalloc((void **)&c, n1 * n2 * sizeof(int));

	// Copy data from host to device
	hipMemcpy(a, a_host, n1 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b, b_host, n2 * sizeof(int), hipMemcpyHostToDevice);

	// Launch the kernel
	multiply<<<1, n1>>>(a, b, c, n1 * n2);

	// Copy data from device to host
	hipMemcpy(c_host, c, n1 * n2 * sizeof(int), hipMemcpyDeviceToHost);

	// Free memory
	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}