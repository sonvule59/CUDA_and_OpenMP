#include "hip/hip_runtime.h"
__global__ void s_e_kernel(int s, int e) {
    if (s * e < 81) {
        __shared__ int shared_s[e];
        
        // Copy s and e into shared memory
        for (int i = 0; i < e; ++i) {
            shared_s[i] = s;
        }
        
        // Update s based on e
        s += (e = e % 9 + 1);
        
        // Check if s is less than 2
        while (s <= 2) {
            std::cout << s << 'x' << e << '=' << s * e << '\n';
            s += 1;
        }
    }
}

int main() {
    int s = 5, e = 3;
    
    // Allocate memory for shared variables
    __shared__ int shared_s[4];
    
    // Initialize shared variables with initial values
    shared_s[0] = s;
    shared_s[1] = e;
    shared_s[2] = s;
    shared_s[3] = e;

    // Launch kernel on GPU
    s_e_kernel<<<1, 4>>>(shared_s, e);

    // Print results
    for (int i = 0; i < 4; ++i) {
        std::cout << shared_s[i] << ' ';
    }

    return 0;
}