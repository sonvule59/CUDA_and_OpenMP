
#include <hip/hip_runtime.h>
#include <stdio.h>
#define FOR(i, a, b) for(int i = (a); i < (b); i++)
#define REP(i, n) for(int i = 0; i < (n); i++)

__global__ void kernel(void){
	FOR(i, 1, 10) FOR(j, 1, 10) printf("%dx%d=%d\n", i, j, i*j);
}

int main(void){
	kernel<<<1, 1>>>();
	hipDeviceSynchronize();
}