
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *x, int *y, int *z, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        z[idx] = x[idx] * y[idx];
    }
}

int main() {
    int x[9], y[9], z[9];
    int n = 9;

    for (int i = 0; i < n; i++) {
        x[i] = i + 1;
        y[i] = i + 1;
    }

    int *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, n * sizeof(int));
    hipMalloc((void **)&d_y, n * sizeof(int));
    hipMalloc((void **)&d_z, n * sizeof(int));

    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(int), hipMemcpyHostToDevice);

    multiply<<<1, n>>>(d_x, d_y, d_z, n);

    hipMemcpy(z, d_z, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d x %d = %d\n", x[i], y[i], z[i]);
    }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}