
#include <hip/hip_runtime.h>
__global__ void add(int* A, int* B, int* C, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        int sum = 0;
        for (int i = 0; i < N; i += blockDim.x * gridDim.x) {
            sum += A[tid] + B[tid];
        }
        C[tid] = sum;
    }
}