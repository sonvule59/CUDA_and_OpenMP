#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

__global__ void multiply(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    int n = 10;
    int *a, *b, *c;
    a = (int *)malloc(n * sizeof(int));
    b = (int *)malloc(n * sizeof(int));
    c = (int *)malloc(n * sizeof(int));

    // Initialize a and b arrays
    for (int i = 0; i < n; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    // Allocate memory on the GPU
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    multiply<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy data from device to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print the result
    for (int i = 0; i < n; i++) {
        cout << a[i