
#include <hip/hip_runtime.h>
__global__ void printMatrix(int matrix[9][9], int a, int b)
{
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadID < 9){
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        int col = threadIdx.z + blockIdx.z * blockDim.z;

        // Check if the element is within range
        if(a <= row && row < 9 && b <= col && col < 9){
            printf("%dx%d=%d\n", matrix[row][col], matrix[a][b], matrix[a * b]);
        }
    }
}