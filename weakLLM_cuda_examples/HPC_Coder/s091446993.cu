#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#define INT(name) int name; scanf("%d",&name);
#define VINT(name,n) vector<int> name(n); for(int i = 0;i < n;i++) scanf("%d",&name[i]);

using namespace std;

__global__ void print_multiplication_table(int *table, int n){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < n){
		for(int i = 1;i <= 9;i++){
			printf("%dx%d=%d\n",i,table[idx],i*table[idx]);
		}
	}
}

int main(int argc, char const *argv[]){
	int n;
	scanf("%d",&n);
	VINT(table,n);
	print_multiplication_table<<<(n+255)/256,256>>>(table,n);
	hipDeviceSynchronize();
	return 0;
}