
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_multiplication_table(int start, int end)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= start && i <= end)
    {
        for (int j = 1; j < 10; j++)
        {
            printf("%dx%d=%d\n", i, j, i * j);
        }
    }
}

int main()
{
    int start = 1;
    int end = 10;
    int blockSize = 256;
    int numBlocks = (end - start + 1 + blockSize - 1) / blockSize;

    print_multiplication_table<<<numBlocks, blockSize>>>(start, end);

    hipDeviceSynchronize();

    return 0;
}