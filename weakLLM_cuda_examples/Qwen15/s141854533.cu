
#include <hip/hip_runtime.h>
__global__ void addSum(int* a, int* b, int* c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < 10) {
        c[tid] = a[tid] + b[tid];
    }
}