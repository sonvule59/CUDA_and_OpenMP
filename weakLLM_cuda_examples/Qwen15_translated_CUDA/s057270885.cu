#include "hip/hip_runtime.h"
__global__ void solvex(int x, int y) {
    if (x > 9) return; // Check if x is greater than 9 before proceeding with the calculation
    printf("dx%d%d=%d\n", y, x, x * y); // Print the result for x = y + 1
    solvex(x + 1, y); // Call the function recursively on x + 1 and y
}

__global__ void solvey(int y) {
    if (y > 9) return; // Check if y is greater than 9 before proceeding with the calculation
    solvex(1, y); // Call the function recursively on 1 and y
    solvey(y + 1); // Call the function recursively on y + 1 and y
}

int main() {
    solvey(1); // Solve y = 1 first
    return 0;
}