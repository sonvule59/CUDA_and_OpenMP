
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int ans) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < 9 && n < 9) {
		ans = (n + 1) * (i + 1);
		printf("%d x %d = %d\n", i + 1, n + 1, ans);
	}

}

int main() {

	int ans = 0;

	dim3 threadsPerBlock(9, 9);
	dim3 numBlocks(1, 1);

	print_table<<<numBlocks, threadsPerBlock>>>(ans);

	hipDeviceSynchronize();

	return 0;

}