
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(uint8_t *a, uint8_t *b, uint8_t *c, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        for (int i = 0; i < n; i++)
        {
            c[idx * n + i] = a[idx * n + i] * b[i * n + idx];
        }
    }
}

int main()
{
    uint8_t *a, *b, *c;
    int n = 10;
    a = (uint8_t *)malloc(n * n * sizeof(uint8_t));
    b = (uint8_t *)malloc(n * n * sizeof(uint8_t));
    c = (uint8_t *)malloc(n * n * sizeof(uint8_t));

    // Initialize a and b

    multiply<<<1, n>>>(a, b, c, n);

    // Print the result

    free(a);
    free(b);
    free(c);
    return 0;
}