#include "hip/hip_runtime.h"
#pragma kernel "kernels/cuda_code.cpp"
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < a.size() && tid < b.size())
    {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void sub(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < a.size() && tid < b.size())
    {
        c[tid] = a[tid] - b[tid];
    }
}

__global__ void mul(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < a.size() && tid < b.size())
    {
        c[tid] = a[tid] * b[tid];
    }
}

__global__ void div(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < a.size() && tid < b.size())
    {
        c[tid] /= b[tid];
    }
}

void addKernel(int *a, int *b, int *c)
{
    __shared__ int sharedData[N];
    int threadIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int blockDimX = blockDim.x / numBlocks;
    int blockDimY = blockDim.y / numBlocks;

    // Add data
    sharedData[threadIdx] = a[blockDimX * blockDimY];
    sharedData[threadIdx + 1] = b[blockDimX * blockDimY];

    // Subtract data
    sharedData[threadIdx] -= a[blockDimX * blockDimY];
    sharedData[threadIdx + 2] -= b[blockDimX * blockDimY];

    // Multiply data
    sharedData[threadIdx] *= a[blockDimX * blockDimY];
    sharedData[threadIdx + 3] *= b[blockDimX * blockDimY];

    // Divide data
    sharedData[threadIdx] /= b[blockDimX * blockDimY];
    sharedData[threadIdx + 4] /= a[blockDimX * blockDimY];

    // Store result
    c[blockDimX * blockDimY] = sharedData[threadIdx];
}

void subKernel(int *a, int *b, int *c)
{
    __