#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 9

__global__ void print_multiplication_table(int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <= N && j <= N) {
        printf("%dx%d=%d\n", i, j, i*j);
    }
}

int main() {
    int N = 9;
    int threadsPerBlock = 16;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    print_multiplication_table<<<blocksPerGrid, threadsPerBlock>>>(N);

    return 0;
}