
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void multiply(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    int a[10], b[10], c[10];
    int n = 10;

    for (int i = 0; i < n; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    multiply<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d x %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}