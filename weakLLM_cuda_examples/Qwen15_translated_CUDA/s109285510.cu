
#include <hip/hip_runtime.h>
__global__ void matrix_multiplication_kernel(float* A, float* B, int m, int n, float* C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m && i < n) {
        float sum = 0.0f;
        
        for (int j = 0; j < n; ++j) {
            sum += A[i*m+j] * B[j];
        }
        
        C[i*m+n] = sum;
    }
}