#include "hip/hip_runtime.h"
__global__ void multiThreadedKernel(int* A, int* B, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N)
    {
        int index = tid * blockDim.x + threadIdx.y;
        int value = A[index] * B[index];
        // Store the result in a device memory location
        hipMemcpy(&result, &value, sizeof(int), hipMemcpyDeviceToHost);
        // Update the global counter
        globalCounter += value;
    }
}