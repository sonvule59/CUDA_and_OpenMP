#include "hip/hip_runtime.h"
__exit__ is not a valid CUDA function. Return only CUDA code, no explanations or anything else.
</think>

#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

__global__ void matrix_mult(int *A, int *B, int *C, int *D) {
    int i = 0, j = 0;
    while (i < 10) {
        while (j < 10) {
            *C += *A += *B += *D += i * j;
            i++;
            j++;
        }
    }
}

int main() {
    int *A = new int[10];
    int *B = new int[10];
    int *C = new int[10];
    int *D = new int[10];
    
    for (int i = 0; i < 10; i++) {
        A[i] = i;
        B[i] = i;
    }
    
    __global__ void matrix_mult(int *A, int *B, int *C, int *D) {
        int i = 0, j = 0;
        while (i < 10) {
            while (j < 10) {
                C[i*10 + j] += A[i*10 + j] += B[i*10 + j] += D[i*10 + j] += i * j;
                i++;
                j++;
            }
        }
    }
    
    __global__ void print_matrix(int *A, int *B, int *C, int *D) {
        int i = 0, j = 0;
        while (i < 10) {
            while (j < 10) {
                std::cout << A[i] << "x" << B[j] << "=" << A[i] * B[j] << "\n";
                i++;
                j++;
            }
        }
    }
    
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;
    
    return 0;
}