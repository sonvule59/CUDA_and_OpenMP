#include "hip/hip_runtime.h"
__global__ void printGrid(int rows, int cols, int value) {
  // Global variables
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.z * blockDim.z + threadIdx.z;

  // Check if the grid is out of bounds
  if (row >= rows || col >= cols || row < 0 || col < 0) {
    return;
  }

  // Calculate the value at the current grid location
  int grid_value = value * row * col;

  // Print the value on the grid
  printf(" %dx%d=%d\n", row, col, grid_value);
}

int main() {
  // Define grid dimensions
  int rows = 10;
  int cols = 10;
  int value = 5;

  // Create a CUDA grid
  dim3 threads_per_block(rows / blockDim.x, cols / blockDim.y, 1);
  dim3 grid_size((rows + threads_per_block.x - 1) / threads_per_block.x,
                 (cols + threads_per_block.y - 1) / threads_per_block.y);

  // Allocate memory for the grid
  int grid[rows][cols];
  
  // Copy the grid values from C++ to CUDA
  hipMemcpy(grid, &value, sizeof(int), hipMemcpyHostToDevice);

  // Launch the kernel
  printGrid<<<grid_size,threads_per_block>>>(rows, cols, grid[0]);

  // Read the grid values back from CUDA
  int grid_values[rows][cols];
  hipMemcpy(grid_values, grid, sizeof(int), hipMemcpyDeviceToHost);

  // Display the grid values
  FOR(i,0,rows) {
    FOR(j,0,cols) {
      printf("%dx%d=%d\n", i, j, grid_values[i][j]);
    }
  }

  return 0;
}