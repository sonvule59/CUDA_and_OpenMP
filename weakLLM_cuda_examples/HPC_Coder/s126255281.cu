
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	printf("%d x %d = %d\n", i, j, i * j);
}

int main(void)
{
	kernel<<<1, 1>>>();
	hipDeviceSynchronize();
	return 0;
}