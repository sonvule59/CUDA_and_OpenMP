#include "hip/hip_runtime.h"
__global__ void matrix multiplication_kernel(float *A, float *B, int N, int M) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= N && col >= M) {
        float sum = 0.0f;
        
        // Access elements of A and B on grid threads
        for (int k = 0; k < N; k++) {
            for (int l = 0; l < M; l++) {
                sum += A[row * M + k] * B[l * N + col];
            }
        }

        // Store result in host memory
        A[row * M + col] = sum;
    }
}