#include "hip/hip_runtime.h"
__global__ void add(xpu_t x, ypu_t y) {
    int a = threadIdx.x + blockIdx.x * blockDim.x;
    int b = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (a < 9 && b < 9) {
        x[a] += y[a];
        y[b] += x[b];
    }
}

int main() {
    dim3 threads(2, 2, 1); // grid size of 2x2x1
    dim3 block_size(1, 1, 1); // block size of 1x1x1
    
    xpu_t x[9], y[9];
    
    add<<<threads, block_size>>>(x, y);
    
    for (int i = 0; i < 9; i++) {
        cout << "Element at thread " << i << ": " << x[i] << ' ' << y[i] << endl;
    }
    
    return 0;
}