#include "hip/hip_runtime.h"
__global__ void kernel(int n, int m)
{
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    if(thread_id < n && thread_id < m)
    {
        int x = threadIdx.x;
        int y = threadIdx.y;

        // Calculate product of x and y for each element
        int product = x * y;

        // Store product in local memory
        int global_product = product;

        // Update global product on all threads
        global_product += n * m;
        
        // Write global product to output device
        __syncthreads();
        global_product /= blockDim.x * blockDim.y;
        hipMemcpy(&output[n*m], &global_product, sizeof(int), hipMemcpyDeviceToHost);

        // Disable synchronization after writing global product to output device
        __syncthreads();

        // Print result
        printf("x = %d, y = %d, Product = %d\n", x, y, product);
    }
}