
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int a, int b)
{
    printf("%d x %d = %d\n", a, b, a*b);
}

int main()
{
    int a, b;
    for(a = 1; a < 10; a++)
        for(b = 1; b < 10; b++)
            print_table<<<1,1>>>(a, b);
    hipDeviceSynchronize();
    return 0;
}