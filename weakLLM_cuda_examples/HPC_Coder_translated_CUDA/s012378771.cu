
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int start, int end) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= start && idx <= end) {
        for (int i = 1; i <= 9; i++) {
            printf("%d x %d = %d\n", idx, i, idx * i);
        }
    }
}

int main() {
    int start = 1;
    int end = 9;
    int threads_per_block = 256;
    int blocks_per_grid = (end - start + 1 + threads_per_block - 1) / threads_per_block;
    print_table<<<blocks_per_grid, threads_per_block>>>(start, end);
    hipDeviceSynchronize();
    return 0;
}