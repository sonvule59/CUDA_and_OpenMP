
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int start, int end) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= start && idx <= end) {
        for (int i = 1; i <= 9; i++) {
            printf("%d x %d = %d\n", idx, i, idx * i);
        }
    }
}

int main() {
    int start = 1;
    int end = 9;
    int threadsPerBlock = 256;
    int blocksPerGrid = (end - start + 1 + threadsPerBlock - 1) / threadsPerBlock;
    print_table<<<blocksPerGrid, threadsPerBlock>>>(start, end);
    hipDeviceSynchronize();
    return 0;
}