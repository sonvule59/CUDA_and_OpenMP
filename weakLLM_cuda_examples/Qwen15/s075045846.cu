#include "hip/hip_runtime.h"
__global__ void print_matrix(int matrix[2][2], int minNum, int maxNum) {
  // Get thread ID
  int threadIdx = blockIdx.x * blockDim.x + threadIdx.y;

  // Check if thread index is within range
  if (threadIdx < minNum || threadIdx > maxNum) {
    return;
  }

  // Access element at row and column indices
  int row = threadIdx.x;
  int col = threadIdx.y;

  // Calculate product of row and column elements
  int product = matrix[row][col];

  // Print element on screen
  std::cout << product << " ";
}