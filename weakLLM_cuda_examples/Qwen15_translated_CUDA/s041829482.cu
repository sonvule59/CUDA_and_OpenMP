
#include <hip/hip_runtime.h>
__global__ void matrix_multiplicationKernel(int n, int m, int* A, int* B, int* C) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(thread_id < n && thread_id < m){
        // Check if both threads have valid indices
        if(A[thread_id] != 0 && B[thread_id] != 0){
            // Calculate product of elements
            int product = A[thread_id] * B[thread_id];
            
            // Store result in local memory
            C[thread_id] = product;
        }
    }
}