#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *a, int *b) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 100) {
		a[idx] = idx;
		b[idx] = idx * 2;
	}
}

int main() {
	int *a, *b;
	int size = 100 * sizeof(int);
	a = (int *)malloc(size);
	b = (int *)malloc(size);

	kernel<<<1, 100>>>(a, b);

	REP(i, 100) {
		printf("%d x %d = %d\n", i, i, a[i] * b[i]);
	}

	free(a);
	free(b);

	return 0;
}