#include "hip/hip_runtime.h"
__global__ void printGrid(int n, int m) {
    // Global variables
    int grid[n][m];
    
    // Local variables
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Iterate over rows
    if(row < n && col < m) {
        grid[row][col] = i+1;
        
        // Check if it's a prime number
        if(isPrime(grid[row][col])) {
            cout << grid[row][col] << " ";
        }
    }
}