#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_runtime_gemm.h>

__global__ void exprKernel(int x, int y) {
    // CUDA kernel implementation
    int threadIdx = threadIdx.x;
    int blockIdx = blockIdx.x;
    
    if(threadIdx<9 && blockIdx<9){
        // Calculate product of x and y on a grid of size 9x9
        int product = x*x + y*y;
        
        // Store the result in a global variable
        string result = expr(product, product);
        
        // Write the result to a file
        ofstream out("result.txt");
        out << result << endl;
    }
}

int main() {
    hipSetDevice(0); // Set device to GPU
    hipMalloc(&deviceMemory, sizeof(string)); // Allocate memory for result variable
    
    // Global variables declaration
    string resultVariable = "";
    
    // Loop over all elements in the set
    for(auto it = c.begin(); it != c.end(); ++it) {
        // Copy element to device memory
        hipMemcpy(deviceMemory, *it, sizeof(string), hipMemcpyHostToDevice);
        
        // Call kernel function
        exprKernel<<<1, 1>>>(deviceMemory, deviceMemory);
        
        // Copy result back to host memory
        hipMemcpy(resultVariable, deviceMemory, sizeof(string), hipMemcpyDeviceToHost);
        
        // Free device memory
        hipFree(deviceMemory);
    }
    
    // Print result variable
    cout << resultVariable << endl;
    
    // Release device resources
    cudaFreeDevice();
    
    return 0;
}