#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <vector>
#include <sstream>

using namespace std;

__global__ void print_table(string* sv, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		cout << sv[idx] << endl;
	}
}

int main() {
	vector<string> sv;
	for (int i = 1; i <= 9; i++) {
		for (int j = 1; j <= 9; j++) {
			stringstream ss;
			ss << i << 'x' << j << '=' << i * j << '\n';
			string temp = ss.str();
			sv.push_back(temp);
		}
	}

	int size = sv.size();
	string* d_sv;
	hipMalloc((void**)&d_sv, size * sizeof(string));
	hipMemcpy(d_sv, sv.data(), size * sizeof(string), hipMemcpyHostToDevice);

	print_table<<<(size + 255) / 256, 256>>>(d_sv, size);

	hipFree(d_sv);

	getchar();
}