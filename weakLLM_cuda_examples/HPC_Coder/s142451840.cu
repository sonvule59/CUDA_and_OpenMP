
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void kernel(int *x, int *y){
	int i, j;
	for(i=0; i<9; i++){
	y[i]=0;
	++x[0];
	for(j=0; j<9; j++){
	++y[i];
	printf("%dx%d=%d\n",x[0],y[i],x[0]*y[i]);
	};
	};
}

int main(){
	int x=0,y=0,i,j;
	int *x_d, *y_d;
	hipMalloc((void **)&x_d, sizeof(int));
	hipMalloc((void **)&y_d, sizeof(int));
	hipMemcpy(x_d, &x, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(y_d, &y, sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1,1>>>(x_d, y_d);
	hipMemcpy(&x, x_d, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&y, y_d, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(x_d);
	hipFree(y_d);
	return 0;
}