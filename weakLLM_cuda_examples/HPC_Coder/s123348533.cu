
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

#define fi first
#define se second
#define all(r) (r).begin(),(r).end()
#define vmax_v(ary) *max_element(all(ary))
#define vmin_v(ary) *min_element(all(ary))
#define vmax_id(ary) distance((ary).begin(),max_element(all(ary)))
#define vmin_id(ary) distance((ary).begin(),min_element(all(ary)))

using ll = int64_t;
using pll = pair<ll,ll>;
using vl = vector<ll>;

const ll INF=INT64_MAX;
const double EPS=1e-10;
const double PI=acos(-1.0);

void init(){cin.tie(0);ios::sync_with_stdio(false);}
//----------------------------------------------------

__global__ void kernel(int* arr){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < 100){
        arr[idx] = idx;
    }
}

int main(){
    init();

    int* arr;
    hipMallocManaged(&arr, 100 * sizeof(int));

    kernel<<<1,100>>>(arr);

    for(int i=0;i<100;i++){
        cout << arr[i] << endl;
    }

    hipFree(arr);

    return 0;
}