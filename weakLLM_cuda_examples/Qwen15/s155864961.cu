#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void kernel_function(int a, int b) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(thread_id < 1 || thread_id >= 10)
        return;

    int temp = (thread_id - 1) * blockDim.x + threadIdx.x;
    int other_thread_id = thread_id - blockDim.x;
    
    // Calculate product of a and b on other thread
    int product = a * b;
    
    // Store product in local memory
    int local_product = temp * product;
    
    // Write product to device memory
    hipMemcpy(&local_product, &product, sizeof(int), hipMemcpyDeviceToHost);
    
    // Update a and b variables on this thread
    a = a + blockDim.x * gridDim.x;
    b = b + blockDim.x * gridDim.x;
}

int main() {
    int a = 1, b = 2;
    
    // Launch kernel function on GPU
    kernel_function<<<1, 1>>>(a, b);

    // Read result from device memory
    int result = cudaGetDeviceLocalMemSize(0, &local_product);
    
    cout << "Result: " << result << endl;

    return 0;
}