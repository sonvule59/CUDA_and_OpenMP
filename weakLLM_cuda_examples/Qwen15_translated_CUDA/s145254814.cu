#include "hip/hip_runtime.h"
__global__ void kernel(int x, int y, int z)
{
    // Global variables
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int global_x = x + thread_id * blockDim.x;
    int global_y = y + thread_id * blockDim.x;

    // Local variables
    int local_x = global_x / blockDim.x;
    int local_y = global_y / blockDim.x;

    // Perform multiplication
    int result = local_x * local_y;

    // Store result in device memory
    __syncthreads();
    if(thread_id % 2 == 0) {
        device->w[local_y] += result;
    } else {
        device->w[local_y] -= result;
    }
    
    // Update global variable
    global_x = global_x - (thread_id % 2 == 0 ? 1 : -1) * blockDim.x;
    global_y = global_y - (thread_id % 2 == 0 ? 1 : -1) * blockDim.x;
    
    // Print result
    printf("x = %d, y = %d, z = %d\n", global_x, global_y, global_z);
}