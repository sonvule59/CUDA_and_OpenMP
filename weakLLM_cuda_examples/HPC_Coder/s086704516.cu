
#include <hip/hip_runtime.h>
#include <stdio.h>

#define pout(n)  printf ("%d\n", n)

#define rep(i,a,n) for (int i = a;i < n;i++)
#define per(i,n,a) for (int i = n-1;i >= a;i--)

const int d4x[4] = {1, 0, -1, 0};
const int d4y[4] = {0, 1, 0, -1};
const int d8x[8] = { 1,1,0,-1,-1,-1,0,1 };
const int d8y[8] = { 0,1,1,1,0,-1,-1,-1 };
 
typedef long long ll;

__global__ void kernel(int *a, int *b, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    a[i] = i * i;
    b[i] = i * i * i;
  }
}

int main() {
  int *a, *b;
  int n = 10;
  hipMalloc((void **)&a, n * sizeof(int));
  hipMalloc((void **)&b, n * sizeof(int));
  kernel<<<1, n>>>(a, b, n);
  hipDeviceSynchronize();
  rep(i,0,n){
    printf("%dx%d=%d\n",i,i,a[i]);
    printf("%dx%d=%d\n",i,i,b[i]);
  }
  hipFree(a);
  hipFree(b);
  return 0;
}