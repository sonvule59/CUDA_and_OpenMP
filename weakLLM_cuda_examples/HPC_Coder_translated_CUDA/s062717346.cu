#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void print_multiplication_table(int a, int b)
{
	printf("%dx%d=%d\n",a,b,a*b);
}

int main()
{
	int a=1,b=1;
	while(a<10&&b<10)
	{
		print_multiplication_table<<<1,1>>>(a,b);
		b++;
		if(b>9)
		{
			a++;
			b=1;
		}
	}
	hipDeviceSynchronize();
	return 0;
}