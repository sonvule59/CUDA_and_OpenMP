
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *seki) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(i<=9 && k<=9) {
		*seki = i * k;
		printf("%d", i);
		printf("x");
		printf("%d", k);
		printf("=");
		printf("%d\n", *seki);
	}
}

int main() {
	int seki;
	int *d_seki;
	
	// Allocate memory on the GPU
	hipMalloc((void **)&d_seki, sizeof(int));
	
	// Launch the kernel
	kernel<<<dim3(9,9), dim3(1,1)>>>(d_seki);
	
	// Copy the result from the GPU to the CPU
	hipMemcpy(&seki, d_seki, sizeof(int), hipMemcpyDeviceToHost);
	
	// Free memory
	hipFree(d_seki);
	
	return 0;
}