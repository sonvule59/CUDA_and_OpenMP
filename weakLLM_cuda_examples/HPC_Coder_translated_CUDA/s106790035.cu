
#include <hip/hip_runtime.h>
#include<stdio.h>
#define F(i) for(i=1;i<=9;i++)
__global__ void kernel(int *a, int *b){
int i,j;
F(i)F(j)a[i*9+j]=i*j;
}
int main(){
int *a, *b;
int i,j;
a=(int *)malloc(9*9*sizeof(int));
b=(int *)malloc(9*9*sizeof(int));
kernel<<<1,1>>>(a,b);
F(i)F(j)printf("%dx%d=%d\n",i,j,a[i*9+j]);
free(a);
free(b);
return 0;
}