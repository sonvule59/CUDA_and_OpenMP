#include "hip/hip_runtime.h"
__global__ void printMatrix(int i, int j, int value)
{
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id < 10)
    {
        // Access element at row and column index
        int row = i * blockDim.x + threadIdx.x;
        int col = j * blockDim.x + threadIdx.x;
        
        // Store element value in global variable
        global_matrix[row][col] = value;
    }
}