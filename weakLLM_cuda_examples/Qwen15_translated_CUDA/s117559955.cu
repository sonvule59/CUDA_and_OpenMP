#include "hip/hip_runtime.h"
__global__ void matrix multiplicationKernel(float* A, float* B, float* C, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row >= m && col >= n) {
        float sum = 0;
        for(int k = 0; k < n; k++) {
            sum += A[row*m+k] * B[col*n+k];
        }

        C[row*m+n] = sum;
    }
}