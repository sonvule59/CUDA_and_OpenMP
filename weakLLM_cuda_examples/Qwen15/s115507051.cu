#include "hip/hip_runtime.h"
__global__ void printMatrix(int rows, int cols) {
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadID < rows && threadID < cols) {
        printf("%d ", matrix[threadID]);
    }
}