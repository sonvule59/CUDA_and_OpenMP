#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <set>
#include <sstream>
#include <cuda_runtime>

using namespace std;

__global__ void kernel(int x, int y, int z) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;
    if (i == 0 && j == 0 && k == 0) {
        z = x * y * z;
    } else if (i == 0 && j == 0 && k != 0) {
        z = x * y * z;
    } else if (i == 0 && j != 0 && k == 0) {
        z = x * y * z;
    } else if (i == 0 && j != 0 && k != 0) {
        z = x * y * z;
    } else if (i == 1 && j == 0 && k == 0) {
        z = x * y * z;
    } else if (i == 1 && j == 0 && k != 0) {
        z = x * y * z;
    } else if (i == 1 && j != 0 && k == 0) {
        z = x * y * z;
    } else if (i == 1 && j != 0 && k != 0) {
        z = x * y * z;
    } else if (i == 2 && j == 0 && k == 0) {
        z = x * y * z;
    } else if (i == 2 && j == 0 && k != 0) {
        z = x * y * z;
    } else if (i == 2 && j != 0 && k == 0) {
        z = x * y * z;
    } else if (i == 2 && j != 0 && k != 0) {
        z = x * y * z;
    } else if (i == 3 && j == 0 && k == 0) {
        z = x * y * z;
    } else if (i == 3 && j == 0 && k != 0) {
        z = x * y * z;
    } else if (i == 3 && j != 0 && k == 0) {
        z = x * y * z;
    }