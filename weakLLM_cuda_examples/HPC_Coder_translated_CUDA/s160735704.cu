
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__ void multiply(int *a, int *b, int *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

int main() {
    int n = 10;
    int *a, *b, *c;
    a = (int *)malloc(n * sizeof(int));
    b = (int *)malloc(n * sizeof(int));
    c = (int *)malloc(n * sizeof(int));

    // Initialize a and b arrays

    multiply<<<1, n>>>(a, b, c, n);

    // Print the result

    free(a);
    free(b);
    free(c);
    return 0;
}