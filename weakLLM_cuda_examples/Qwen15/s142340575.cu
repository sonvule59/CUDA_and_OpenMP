
#include <hip/hip_runtime.h>
__global__ void printGridKernel(float* grid, int rows, int cols) {
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadID < rows && threadID < cols)
        grid[threadID] = (threadID + 1) * (threadID + 2);
}