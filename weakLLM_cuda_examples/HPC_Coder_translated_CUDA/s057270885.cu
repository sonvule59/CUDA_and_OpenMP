#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void solvex(int x, int y){
	if(x > 9)return ;
	printf("%dx%d=%d\n",y,x,x*y);
	solvex(x+1,y);
}

__global__ void solvey(int y){
	if(y > 9)return ;
	solvex(1, y);
	solvey(y+1);
}

int main(){
	solvey<<<1,1>>>(1);
	hipDeviceSynchronize();
	return 0;
}