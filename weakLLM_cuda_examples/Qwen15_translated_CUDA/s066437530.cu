#include "hip/hip_runtime.h"
__global__ void print_matrix(int i, int j, int n) {
    // Global variables
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.z + blockIdx.z * blockDim.z;

    // Check if the index is within the matrix dimensions
    if (row >= 0 && row < n && col >= 0 && col < n)
    {
        // Calculate the dot product of two elements at this position
        int dot_product = i * j * n;

        // Store the dot product in a local variable
        int element = dot_product / n;

        // Print the element on the screen
        std::cout << "Element " << element << "\n";
    }
}