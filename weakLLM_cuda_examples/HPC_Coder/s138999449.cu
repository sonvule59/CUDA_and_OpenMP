
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void multiply(int *a, int *b, int *c)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int z = threadIdx.z;
	int a_val = a[x];
	int b_val = b[y];
	int c_val = a_val * b_val;
	c[z] = c_val;
}

int main(void)
{
	int a[9][9] = {0};
	int b[9][9] = {0};
	int c[9][9] = {0};
	int *d_a, *d_b, *d_c;
	int size = sizeof(a)/sizeof(a[0]);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	multiply<<<1, size>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	for(int i=0;i<9;i++){
		for(int j=0;j<9;j++){
			printf("%dx%d=%d\n",a[i],b[j],c[i][j]);
		}
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}