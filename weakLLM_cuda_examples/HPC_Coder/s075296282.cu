
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void multiply(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 10) {
		c[idx] = a[idx] * b[idx];
	}
}

int main() {
	int a[10], b[10], c[10];
	for (int i = 0; i < 10; i++) {
		a[i] = i + 1;
		b[i] = i + 1;
	}
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, 10 * sizeof(int));
	hipMalloc((void **)&d_b, 10 * sizeof(int));
	hipMalloc((void **)&d_c, 10 * sizeof(int));
	hipMemcpy(d_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, 10 * sizeof(int), hipMemcpyHostToDevice);
	multiply<<<1, 10>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, 10 * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++) {
		printf("%d x %d = %d\n", a[i], b[i], c[i]);
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}