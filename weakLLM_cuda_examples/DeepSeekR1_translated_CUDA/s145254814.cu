#include "hip/hip_runtime.h"
__global__ i, __global__ j, __global__ i*j;
printf("%dx%d=%d\n", __global__i, __global__j, __global__i*j);
return 0;
</think>

#include <stdio.h>

int main(int argc, char const *argv[])
{
	int i, j;
	for(i=1;i<10;i++) {
		for(j=1;j<10;j++) {
			printf("%dx%d=%d\n", i, j, i*j);
		}
	}
	return 0;
}

__global__ i, __global__ j, __global__ i*j;
printf("%dx%d=%d\n", __global__i, __global__j, __global__i*j);
return 0;