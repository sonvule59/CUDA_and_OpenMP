
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_multiplication_table() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 9 && j < 9) {
        int m = i + 1;
        int n = j + 1;
        printf("%d x %d = %d\n", m, n, m * n);
    }
}

int main() {
    dim3 threadsPerBlock(3, 3);
    dim3 numBlocks(3, 3);  // 3x3 blocks * 3x3 threads = 9x9 grid

    print_multiplication_table<<<numBlocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    return 0;
}
