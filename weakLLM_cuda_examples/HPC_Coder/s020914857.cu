
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int m, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < m) {
    for (int i = 1; i <= n; i++) {
      printf("%d x %d = %d\n", idx, i, idx * i);
    }
  }
}

int main() {
  int m, n;
  m = 1;
  n = 1;

  while (true) {
    if (n == 10) {
      m++;
      n = 1;
    }
    if (m == 10) break;

    print_table<<<1, 1>>>(m, n);
    hipDeviceSynchronize();
    n++;
  }
  return 0;
}