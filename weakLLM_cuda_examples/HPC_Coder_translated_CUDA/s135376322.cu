#include "hip/hip_runtime.h"
#include<iostream>

using namespace std;

__global__ void kernel(int *a, int *b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int kake = i * j;
	a[i] = kake;
	b[j] = kake;
}

int main() {
	int a[10][10];
	int b[10][10];

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((10+threadsPerBlock.x-1)/threadsPerBlock.x, (10+threadsPerBlock.y-1)/threadsPerBlock.y);

	kernel<<<numBlocks, threadsPerBlock>>>(a, b);

	for (int i=0; i<10; i++) {
		for (int j=0; j<10; j++) {
			cout << a[i][j] << "x" << b[j][i] << "=" << a[i][j] * b[j][i] << endl;
		}
	}
	return 0;
}