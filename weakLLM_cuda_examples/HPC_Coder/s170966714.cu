
#include <hip/hip_runtime.h>
#include<stdio.h>
#define REP(i,n) for(int (i)=0;(i)<(int)(n);(i)++)
using namespace std;

__global__ void kernel(int *a, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        REP(j,9)printf("%d x %d = %d\n", idx+1, j+1, (idx+1)*(j+1));
    }
}

int main(){
    int n = 9;
    int *a;
    hipMalloc((void **)&a, n*n*sizeof(int));
    kernel<<<1,n>>>(a, n);
    hipDeviceSynchronize();
    hipFree(a);
    return 0;
}