
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void print_table(int *table) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 100) {
        for (int i = 1; i <= 9; i++) {
            for (int j = 1; j <= 9; j++) {
                printf("%d x %d = %d\n", i, j, i * j);
            }
        }
    }
}

int main() {
    int *table;
    hipMalloc((void **)&table, sizeof(int) * 100);
    print_table<<<1, 100>>>(table);
    hipDeviceSynchronize();
    hipFree(table);
    return 0;
}