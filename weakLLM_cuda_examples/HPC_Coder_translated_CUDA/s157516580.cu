
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_table(int *table) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;
	int l = i * k;
	printf("%d x %d = %d\n", i, k, l);
}

int main() {
	int table[10][10];
	for(int i = 0; i < 10; i++) {
		for(int k = 0; k < 10; k++) {
			table[i][k] = i * k;
		}
	}

	int *d_table;
	hipMalloc((void **)&d_table, 10 * 10 * sizeof(int));
	hipMemcpy(d_table, table, 10 * 10 * sizeof(int), hipMemcpyHostToDevice);

	int threadsPerBlock = 16;
	int blocksPerGrid = (10 * 10 + threadsPerBlock - 1) / threadsPerBlock;
	print_table<<<blocksPerGrid, threadsPerBlock>>>(d_table);

	hipDeviceSynchronize();
	hipFree(d_table);
	return 0;
}