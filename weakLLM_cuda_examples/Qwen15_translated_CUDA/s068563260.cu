
#include <hip/hip_runtime.h>
__global__ void mult_matrix_kernel(int a, int b, int c) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id < a && thread_id < b && thread_id < c)
        c = a*b;
    
    // Write the result to device memory
    __shared__ int shared_result[3];
    shared_result[thread_id] = c;
    
    // Update global variables
    a += blockDim.x * gridDim.x;
    b += blockDim.x * gridDim.x;
}