
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>
#include<vector>
#include<algorithm>
#include<map>
#include<deque>
#include<cstring>
#include<climits>
#include<cassert>
#include<cctype>

#define REP(i,s,n) for(int i=s;i<n;++i)
#define rep(i,n) REP(i,0,n)

using namespace std;

__global__ void kernel(int *a, int *b) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < 9) {
    a[idx] = (idx+1)*(idx+1);
    printf("%d x %d = %d\n", idx+1, idx+1, (idx+1)*(idx+1));
  }
}

int main() {
  int *a, *b;
  hipMalloc((void **)&a, 9*sizeof(int));
  hipMalloc((void **)&b, 9*sizeof(int));

  kernel<<<1,9>>>(a, b);

  hipDeviceSynchronize();

  hipFree(a);
  hipFree(b);

  return 0;
}