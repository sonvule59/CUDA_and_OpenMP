#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void matrixMultiplicationKernel(int rows, int cols, int alpha, int beta, int *A, int *B, int *C) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if(row >= rows && col >= cols) {
    int index = row * cols + col;
    int sum = alpha * A[index] + beta * B[index];

    // Copy the result to the output array
    C[index] = sum;
  }
}

void matrixMultiplication() {
  int rows = 100000; // Replace with your actual number of rows
  int cols = 100000; // Replace with your actual number of columns

  int alpha = 5; // Replace with your desired value of alpha
  int beta = 3; // Replace with your desired value of beta

  int *A = new int[rows * cols]; // Allocate memory for the input matrix A
  int *B = new int[rows * cols]; // Allocate memory for the input matrix B
  int *C = new int[rows * cols]; // Allocate memory for the output matrix C

  // Initialize the matrices with random values
  for(int i=0; i<rows; ++i) {
    for(int j=0; j<cols; ++j) {
      A[i * cols + j] = rand() % 256;
    }
  }

  // Multiply the matrices using the kernel
  matrixMultiplicationKernel(rows, cols, alpha, beta, A, B, C);

  // Print the resulting matrix C
  for(int i=0; i<rows; ++i) {
    for(int j=0; j<cols; ++j) {
      cout << C[i * cols + j] << ' ';
    }
    cout << endl;
  }
}