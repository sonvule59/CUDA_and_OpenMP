#include "hip/hip_runtime.h"
__global__ void kernel(int x, int y) {
    int sum = 0;
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < blockDim.x * gridDim.x; i += blockDim.x * blockDim.y) {
        sum += x * y;
    }
    
    // Store the result in a device variable
    __shared__ int sharedSum __device__(align(64));
    sharedSum[0] = sum;
    
    // Write the result back to host memory
    hipMemcpy(hostSum, sharedSum, sizeof(int), hipMemcpyDeviceToHost);
    
    // Print the result
    std::cout << "Sum of " << x << "x" << y << " on GPU: " << hostSum << std::endl;
}