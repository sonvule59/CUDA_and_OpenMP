#include "hip/hip_runtime.h"
#include <stdio.h>
int s,e;
__global__ void kernel(int *s, int *e) {
    while(s*e<81)s+=(e=e%9+1)<2;
}
int main() {
    int s, e;
    int *d_s, *d_e;
    hipMalloc((void **)&d_s, sizeof(int));
    hipMalloc((void **)&d_e, sizeof(int));
    hipMemcpy(d_s, &s, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_e, &e, sizeof(int), hipMemcpyHostToDevice);
    kernel<<<1,1>>>(d_s, d_e);
    hipMemcpy(&s, d_s, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&e, d_e, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d x %d = %d\n", s, e, s*e);
    hipFree(d_s);
    hipFree(d_e);
    return 0;
}