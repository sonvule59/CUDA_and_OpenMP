
#include <hip/hip_runtime.h>
__global__ void kernel(float *a, float *b, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < n)
    {
        float sum = 0.0f;
        
        for(int i = tid; i < n; i += blockDim.x * gridDim.x)
        {
            sum += a[i] * b[i];
        }
        
        // Store result in host memory
        a[tid] = sum;
    }
}