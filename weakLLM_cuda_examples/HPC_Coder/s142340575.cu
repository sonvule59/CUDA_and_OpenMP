
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void kernel(int *a, int *b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<9 && j<9){
        printf("%dx%d=%d\n",i+1,j+1,(i+1)*(j+1));
    }
}

int main(){
    int *a, *b;
    hipMalloc((void **)&a, sizeof(int));
    hipMalloc((void **)&b, sizeof(int));
    kernel<<<dim3(9,9), dim3(1,1)>>>(a, b);
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    return 0;
}